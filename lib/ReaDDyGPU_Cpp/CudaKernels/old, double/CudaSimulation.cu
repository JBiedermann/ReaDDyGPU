#include "hip/hip_runtime.h"

# include <ReaDDyGPU.hpp>
# include <stdlib.h>
# include <stdio.h>
# include <time.h>
# include <math.h>
# include <vector>
# include <hip/hip_runtime.h>
# include <hiprand.h>
# include <hip/hip_runtime.h>
# include <hiprand/hiprand_kernel.h>
# include <sm_11_atomic_functions.h>
#include <initializer_list>

/// //////////////////////////////////////////////////////////////////////////////////////////
/// /
/// TODO:
///         - data structures
///         - dynamic arrays (amortized linear runtime) +sort?
///         - periodic boundaries
///         - lattice + neighbor lists
///         - usage of shared mem
/// /
/// //////////////////////////////////////////////////////////////////////////////////////////


__global__ void update(double* cudaCoords, double* cudaForces, int* cudaTypes, double* cudaD, int * cudaNeighborList, int * cudaNeighborListBegins, double * cudaBoxSize, int * cudaSemaphore, hiprandState* globalRandStates, double dt, int numberParticles, double KB, double T, double maxCutoff, int * latticeSize);
__global__ void orderOne(double* cudaCoords, double* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, double * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderOnePotentialsMatrix, CudaOrderOnePotential * cudaCudaOrderOnePotentials, int numberOfOrderOnePotentials, int numberOfParticleTypes, double * cudaParticleRadiiMatrix);
__global__ void orderTwo(double* cudaCoords, double* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, double * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderTwoPotentialsMatrix, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfOrderTwoPotentials, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix);
__global__ void groups(double* cudaCoords, double* cudaForces, int* cudaTypes, int numberParticles, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix, CudaOrderTwoPotential * cudaCudaGroupPotentials, GroupPart * cudaIndividualGroups, int numberOfIndividualGroups, double * cudaBoxSize);
__device__ void getNeighbors(int particleNumber, int * todo, double* cudaCoords, int * cudaLatticeSize, double * cudaBoxSize, int maxCutoff);
__device__ void calculateOrderTwoPotential(int particleNumber, int interactingParticle, int orderTwoPotentialNr, double r, double* cudaCoords, double* cudaForces, int* cudaTypes, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix);
__global__ void calculateRDF(int * cudaRDFMatrix, double* cudaCoords, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, double * cudaBoxSize, int numberParticles, int maxCutoff, int numberOfParticleTypes, int numberOfBins);
__global__ void setup_kernel ( hiprandState * state, unsigned long seed, int n );

__global__ void warpOrderTwo(int warpsize, double* cudaCoords, double* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, double * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderTwoPotentialsMatrix, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfOrderTwoPotentials, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix);
__device__ void calculateOrderTwoPotential(double * particleCoord, double * particleForce, int particleType, double * interactingParticleCoord, double * interactingParticleForce, int interactingParticleType, int orderTwoPotentialNr, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, double * cudaBoxSize, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix);

CudaSimulation::CudaSimulation(Simulation* simulation){

    this->simulation = simulation;

}

/*__device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old); return __longlong_as_double(old);
}*/
__device__ double atomicAdd(double* address, double val) {
    unsigned int* address_as_ui = (unsigned int*)address;
    unsigned int old = *address_as_ui, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ui, assumed, __float_as_int(val + __int_as_float(assumed)));
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old); return __int_as_float(old);
}
///Read more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#ixzz3GDRkYOfX


CudaOrderOnePotential toCudaOrderOnePotential(OrderOnePotential* orderOnePotential){
    CudaOrderOnePotential cudaOrderOnePotential = CudaOrderOnePotential();
    if(orderOnePotential->type.compare("DISK")==0){
        DiskPotential * diskPotential = reinterpret_cast<DiskPotential*>(orderOnePotential);
        cudaOrderOnePotential.type=diskPotential->typeID;
        cudaOrderOnePotential.subtype=diskPotential->subtypeID;
        cudaOrderOnePotential.forceConst=diskPotential->forceConst;
        std::copy ( diskPotential->center, diskPotential->center+3, cudaOrderOnePotential.origin );
        std::copy ( diskPotential->normal, diskPotential->normal+3, cudaOrderOnePotential.normal );
        cudaOrderOnePotential.radius=diskPotential->radius;
    }
    else if(orderOnePotential->type.compare("CYLINDER")==0){
        CylinderPotential * cylinderPotential = reinterpret_cast<CylinderPotential*>(orderOnePotential);
        cudaOrderOnePotential.type=cylinderPotential->typeID;
        cudaOrderOnePotential.subtype=cylinderPotential->subtypeID;
        cudaOrderOnePotential.forceConst=cylinderPotential->forceConst;
        std::copy ( cylinderPotential->center, cylinderPotential->center+3, cudaOrderOnePotential.origin );
        std::copy ( cylinderPotential->normal, cylinderPotential->normal+3, cudaOrderOnePotential.normal );
        cudaOrderOnePotential.radius=cylinderPotential->radius;
        cudaOrderOnePotential.height=cylinderPotential->height;
    }
    else if(orderOnePotential->type.compare("SPHERE")==0){
        SpherePotential * spherePotential = reinterpret_cast<SpherePotential*>(orderOnePotential);
        cudaOrderOnePotential.type=spherePotential->typeID;
        cudaOrderOnePotential.subtype=spherePotential->subtypeID;
        cudaOrderOnePotential.forceConst=spherePotential->forceConst;
        std::copy ( spherePotential->center, spherePotential->center+3, cudaOrderOnePotential.origin );
        cudaOrderOnePotential.radius=spherePotential->radius;
    }
    else if(orderOnePotential->type.compare("BOX")==0 || orderOnePotential->type.compare("CUBE")==0){
        BoxPotential * boxPotential = reinterpret_cast<BoxPotential*>(orderOnePotential);
        cudaOrderOnePotential.type=boxPotential->typeID;
        cudaOrderOnePotential.subtype=boxPotential->subtypeID;
        cudaOrderOnePotential.forceConst=boxPotential->forceConst;
        std::copy ( boxPotential->origin, boxPotential->origin+3, cudaOrderOnePotential.origin );
        std::copy ( boxPotential->extension, boxPotential->extension+3, cudaOrderOnePotential.extension );
    }
    return cudaOrderOnePotential;
}

CudaOrderTwoPotential toCudaOrderTwoPotential(OrderTwoPotential* orderTwoPotential){
    CudaOrderTwoPotential cudaOrderTwoPotential = CudaOrderTwoPotential();
    if(orderTwoPotential->type.compare("HARMONIC")==0){
        HarmonicPotential * harmonicPotential = reinterpret_cast<HarmonicPotential*>(orderTwoPotential);
        cudaOrderTwoPotential.type=harmonicPotential->typeID;
        cudaOrderTwoPotential.subtype=harmonicPotential->subtypeID;
        cudaOrderTwoPotential.forceConst = harmonicPotential->forceConst;
    }
    else if(orderTwoPotential->type.compare("WEAK_INTERACTION_HARMONIC")==0){
        WeakInteractionHarmonicPotential * weakInteractionHarmonicPotential = reinterpret_cast<WeakInteractionHarmonicPotential*>(orderTwoPotential);
        cudaOrderTwoPotential.type=weakInteractionHarmonicPotential->typeID;
        cudaOrderTwoPotential.subtype=weakInteractionHarmonicPotential->subtypeID;
        cudaOrderTwoPotential.forceConst=weakInteractionHarmonicPotential->forceConst;
        cudaOrderTwoPotential.length=weakInteractionHarmonicPotential->length;
        cudaOrderTwoPotential.depth=weakInteractionHarmonicPotential->depth;
    }
    return cudaOrderTwoPotential;
}

int CudaSimulation::initialize(){

    int numberOfCudaDevices = 0;
    hipGetDeviceCount(&numberOfCudaDevices);

    if(numberOfCudaDevices==0){
        cout << "no cuda device availible" << endl;
        return 1;
    }
    if(simulation->testmode)
        cout << endl << endl << numberOfCudaDevices << " cuda devices found" << endl << endl;

    for(int i=0; i<numberOfCudaDevices; ++i){

        hipSetDevice(i);
        struct hipDeviceProp_t prop;

        hipGetDeviceProperties(&prop, i);
        if(simulation->testmode){
            cout << prop.name << endl;
            cout << "compute capability: " << prop.major << "." << prop.minor << endl;

            cout << "total global Memory: " << (float)prop.totalGlobalMem/1024.0f/1024.0f/1024.0f << "GB" << endl;
            cout << "shared memory per block: " << (float)prop.sharedMemPerBlock/1024.0f << "KB" << endl;
            cout << "total constant memory: " << (float)prop.totalConstMem/1024.0f << "KB" << endl;
            cout << "memory clock rate: " << prop.memoryClockRate << "Hz" << endl;
            cout << "memory bus width: " << prop.memoryBusWidth << "bits" << endl;

            cout << "multi processors: " << prop.multiProcessorCount << endl;
            cout << "clock rate: " << prop.clockRate << "Hz" << endl;

            cout << "warpsize: " << prop.warpSize << endl;
            cout << "max threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << endl;
            cout << "max threads dim: " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << endl;
            cout << "max grid size: " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << endl;
            cout << endl;
        }
    }

    /// ////////////////////////////////////////////////////////////////////////
    cudaDevice = 3;
    numberOfThreads = 128;
    /// ////////////////////////////////////////////////////////////////////////

    hipSetDevice(cudaDevice);

    gridSize = (simulation->numberParticles/numberOfThreads)+1;
    blockSize = numberOfThreads;
    //gridSize = 10;
    //blockSize = 10;

    if(simulation->testmode)
        cout << "use " << gridSize<< " blocks (grid size) and " << blockSize << " threads (block size) each" << endl;

    maxCutoff = simulation->maxCutoff;
    boxSize = simulation->latticeBounds;

if(simulation->testmode)
    cout << "initialize cuda random variables" << endl;

    /// initialize cuRand
    hipMalloc ( (void**)&globalRandStates, simulation->numberParticles * sizeof( hiprandState ) );
    /// setup seeds
    setup_kernel <<< gridSize, blockSize >>> ( globalRandStates, time(NULL), simulation->numberParticles );


if(simulation->testmode)
    cout << "initialize several cuda arrays" << endl;

    /// Coords
    hipMalloc((void**)&cudaCoords,( simulation->numberParticles * 3 * sizeof ( double ) ));
    copyPosToDevice();

    /// Forces
    hipMalloc((void**)&cudaForces,( simulation->numberParticles * 3 * sizeof ( double ) ));
    hipMemset( cudaForces,(double)0, ( simulation->numberParticles * 3 * sizeof ( double ) ));

    /// Diffusion const.
    double * hostDiffConst;
    hostDiffConst = new double[simulation->particleTypes.size()];
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        hostDiffConst[i]=simulation->particleTypes[i].D;
    }
    hipMalloc((void**)&cudaD,( simulation->particleTypes.size() * sizeof ( double ) ));
    hipMemcpy(cudaD, hostDiffConst, ( simulation->particleTypes.size() * sizeof ( double ) ), hipMemcpyHostToDevice);

    /// types
    hipMalloc((void**)&cudaTypes,( simulation->numberParticles * sizeof ( int ) ));
    hipMemcpy(cudaTypes, simulation->types, ( simulation->numberParticles * sizeof ( int ) ), hipMemcpyHostToDevice);

    if(createNeighborList()!=0){
        cout <<"neigborlist building problem" << endl;
        return 1;
    }

    hipMemcpy(cudaNeighborList, hostNeighborList, ( simulation->numberParticles * 2 * sizeof ( int ) ), hipMemcpyHostToDevice);
    hipMemcpy(cudaNeighborListBegins, hostNeighborListBegins, ( numberOfLatticeFields * sizeof ( int ) ), hipMemcpyHostToDevice);

    hipMalloc ((void**)&cudaBoxSize, ( 6 * sizeof ( double ) ));
    hipMemcpy( cudaBoxSize, boxSize, ( 6 * sizeof ( double ) ), hipMemcpyHostToDevice);
    hipMalloc (   (void**)&cudaLatticeSize, ( 3 * sizeof ( int ) ));
    hipMemcpy(cudaLatticeSize, latticeSize, ( 3 * sizeof ( int ) ), hipMemcpyHostToDevice);

    /// cudaSemaphores for the lattice fields
    hipMalloc((void**)&cudaSemaphore,( numberOfLatticeFields * sizeof ( int ) ));
    hipMemset( cudaSemaphore,(int)0, ( numberOfLatticeFields * sizeof ( int ) ));

if(simulation->testmode)
    cout << "initialize cuda order one potentials" << endl;

    /// Matrix for order one potentials = matrix[pot][types] = matrix[simulation->orderOnePotentials.size()][simulation->particleTypes.size()]
    int orderOnePotentialsMatrixSize = simulation->particleTypes.size() * simulation->orderOnePotentials.size();
    hostOrderOnePotentialsMatrix = new int[orderOnePotentialsMatrixSize];
    for(int i=0; i<simulation->orderOnePotentials.size(); ++i){
        for(int j=0; j<simulation->particleTypes.size(); ++j){
            hostOrderOnePotentialsMatrix[i*simulation->particleTypes.size()+j]=0;
        }
        for(int j=0; j<simulation->orderOnePotentials[i]->affectedParticleTypeIds.size(); ++j){
            hostOrderOnePotentialsMatrix[i*simulation->particleTypes.size()+simulation->orderOnePotentials[i]->affectedParticleTypeIds[j]]=1;
        }
    }
    hipMalloc((void**)&cudaOrderOnePotentialsMatrix,( orderOnePotentialsMatrixSize * sizeof ( int ) ));
    hipMemcpy(cudaOrderOnePotentialsMatrix, hostOrderOnePotentialsMatrix, ( orderOnePotentialsMatrixSize * sizeof ( int ) ), hipMemcpyHostToDevice);

    /// create cuda order one pot
    hostCudaOrderOnePotentials = new CudaOrderOnePotential[simulation->orderOnePotentials.size()];
    for(int i=0; i<simulation->orderOnePotentials.size(); ++i){
        hostCudaOrderOnePotentials[i] = toCudaOrderOnePotential(simulation->orderOnePotentials[i]);
    }
    hipMalloc((void**)&cudaCudaOrderOnePotentials,( simulation->orderOnePotentials.size() * sizeof ( CudaOrderOnePotential ) ));
    hipMemcpy(cudaCudaOrderOnePotentials, hostCudaOrderOnePotentials, ( simulation->orderOnePotentials.size() * sizeof ( CudaOrderOnePotential ) ), hipMemcpyHostToDevice);

if(simulation->testmode)
    cout << "initialize cuda order two potentials" << endl;

    /// Lookup for order two potentials
    int numberOfParticleTypes = simulation->particleTypes.size();
    int orderTwoPotentialsMatrixSize = numberOfParticleTypes * numberOfParticleTypes * simulation->orderTwoPotentials.size();
    hostOrderTwoPotentialsMatrix = new int[orderTwoPotentialsMatrixSize];
    for(int numberParticleTypes1=0; numberParticleTypes1<numberOfParticleTypes; ++numberParticleTypes1){
        //cout << numberParticleTypes1 << endl;
        for(int numberParticleTypes2=0; numberParticleTypes2<numberOfParticleTypes; ++numberParticleTypes2){
            //cout << " " << numberParticleTypes2 << endl;
            for(int numberOrderTwoPotentials=0; numberOrderTwoPotentials<simulation->orderTwoPotentials.size(); ++numberOrderTwoPotentials){
                //cout << "  " << numberOrderTwoPotentials << endl;
                hostOrderTwoPotentialsMatrix[numberParticleTypes1*numberOfParticleTypes+numberParticleTypes2*numberOfParticleTypes+numberOrderTwoPotentials]=0;
            }
        }
    }
    for(int orderTwoPotential=0; orderTwoPotential<simulation->orderTwoPotentials.size(); ++orderTwoPotential){
        for(int i=0; i<simulation->orderTwoPotentials[orderTwoPotential]->affectedParticleTypeIdPairs.size(); i+=2){
            //cout << " " << i << endl;
            int particleType1= simulation->orderTwoPotentials[orderTwoPotential]->affectedParticleTypeIdPairs[i];
            int particleType2= simulation->orderTwoPotentials[orderTwoPotential]->affectedParticleTypeIdPairs[i+1];
            hostOrderTwoPotentialsMatrix[particleType1*numberOfParticleTypes+particleType2*numberOfParticleTypes+orderTwoPotential]=1;
            hostOrderTwoPotentialsMatrix[particleType2*numberOfParticleTypes+particleType1*numberOfParticleTypes+orderTwoPotential]=1;
        }
    }

    hipMalloc((void**)&cudaOrderTwoPotentialsMatrix,( orderTwoPotentialsMatrixSize * sizeof ( int ) ));
    hipMemcpy(cudaOrderTwoPotentialsMatrix, hostOrderTwoPotentialsMatrix, ( orderTwoPotentialsMatrixSize * sizeof ( int ) ), hipMemcpyHostToDevice);

    /// create cuda order two pot
    hostCudaOrderTwoPotentials = new CudaOrderTwoPotential[simulation->orderTwoPotentials.size()];
    for(int i=0; i<simulation->orderTwoPotentials.size(); ++i){
        hostCudaOrderTwoPotentials[i] = toCudaOrderTwoPotential(simulation->orderTwoPotentials[i]);
    }
    hipMalloc((void**)&cudaCudaOrderTwoPotentials,( simulation->orderTwoPotentials.size() * sizeof ( CudaOrderTwoPotential ) ));
    hipMemcpy(cudaCudaOrderTwoPotentials, hostCudaOrderTwoPotentials, ( simulation->orderTwoPotentials.size() * sizeof ( CudaOrderTwoPotential ) ), hipMemcpyHostToDevice);

if(simulation->testmode)
    cout << "initialize cuda particle radii and collision radii matrix" << endl;

    /// create cuda paritcle radii matix (for order one ptoentials
    hostParticleRadiiMatrix = new double[simulation->particleTypes.size()];
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        hostParticleRadiiMatrix[i]=simulation->particleTypes[i].defaultRadius;
        //cout << hostParticleRadiiMatrix[i] << endl;
    }
    hipMalloc((void**)&cudaParticleRadiiMatrix,( (simulation->particleTypes.size()) * sizeof ( double ) ));
    hipMemcpy(cudaParticleRadiiMatrix, hostParticleRadiiMatrix, ( simulation->particleTypes.size() * sizeof ( double ) ), hipMemcpyHostToDevice);

    /// create cuda collision radii matix for order two potentials
    hostCollisionRadiiMatrix = new double[(simulation->particleTypes.size())*simulation->particleTypes.size()];
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=0; j<simulation->particleTypes.size(); ++j){
            hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j] = simulation->particleTypes[i].radiiMatrix[j];
        }
        for(int j=0; j<simulation->particleTypes.size(); ++j){
            hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j] = hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j]== 0 ? simulation->particleTypes[i].defaultRadius : hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j];
        }
    }
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=i; j<simulation->particleTypes.size(); ++j){
            double x = hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j]+hostCollisionRadiiMatrix[j*simulation->particleTypes.size()+i];
            hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j] = x;
            hostCollisionRadiiMatrix[j*simulation->particleTypes.size()+i] = x;
        }
    }
    /*for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=0; j<simulation->particleTypes.size(); ++j){
            cout << hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j] << " " ;
        }
        cout << endl;
    }*/
    hipMalloc((void**)&cudaCollisionRadiiMatrix,( (simulation->particleTypes.size())*simulation->particleTypes.size() * sizeof ( double ) ));
    hipMemcpy(cudaCollisionRadiiMatrix, hostCollisionRadiiMatrix, ( (simulation->particleTypes.size())*simulation->particleTypes.size() * sizeof ( double ) ), hipMemcpyHostToDevice);

if(simulation->testmode)
    cout << "initialize cuda group potentials" << endl;

    /// create cuda group potentials
    int numberOfGroupPotentials=0;
    for(int i=0; i<simulation->groups.size(); ++i){
        numberOfGroupPotentials += simulation->groups[i].potentials.size();
    }
    hostCudaGroupPotentials = new CudaOrderTwoPotential[numberOfGroupPotentials];
    int continuousForceNumber=0;
    for(int i=0; i<simulation->groups.size(); ++i){
        for(int j=0; j<simulation->groups[i].potentials.size(); ++j, ++continuousForceNumber){
            hostCudaGroupPotentials[continuousForceNumber] = toCudaOrderTwoPotential(simulation->groups[i].potentials[j]);
//cout <<"group "  << i << " force "<< hostCudaGroupPotentials[continuousForceNumber].type << endl;
        }
    }
    hipMalloc((void**)&cudaCudaGroupPotentials,( numberOfGroupPotentials * sizeof ( CudaOrderTwoPotential ) ));
    hipMemcpy(cudaCudaGroupPotentials, hostCudaGroupPotentials, ( numberOfGroupPotentials * sizeof ( CudaOrderTwoPotential ) ), hipMemcpyHostToDevice);

    /// list of particle pairs with potential calculations from a group
    vector<GroupPart> groupParts = vector<GroupPart>();
    continuousForceNumber=0;
    for(int i=0; i<simulation->groups.size(); ++i){
        for(int j=0; j<simulation->groups[i].individualGroups.size(); ++j){
            //for(int k=0; k<simulation->groups[i].individualGroups[j].size(); ++k){
            /// TODO: later for more than two!!
                GroupPart groupPart;
                groupPart.particle1=simulation->groups[i].individualGroups[j][0];
                groupPart.particle2=simulation->groups[i].individualGroups[j][1];
                groupPart.groupPot=continuousForceNumber;
                groupParts.push_back(groupPart);
//cout << simulation->groups[i].individualGroups[j][0] << " " <<simulation->groups[i].individualGroups[j][1] <<" " << continuousForceNumber<< endl;
            //}
            //++continuousForceNumber;
        }
    }
    hostIndividualGroups = &groupParts[0];
    numberOfIndividualGroups = groupParts.size();
    /*for(int i=0; i<numberOfIndividualGroups; ++i){
        cout << hostIndividualGroups[i].particle1 << " " << hostIndividualGroups[i].particle2 << " " << hostIndividualGroups[i].groupPot << " " << hostCudaGroupPotentials[hostIndividualGroups[i].groupPot].forceConst << " " << hostCudaGroupPotentials[hostIndividualGroups[i].groupPot].type << " " << hostCudaGroupPotentials[hostIndividualGroups[i].groupPot].subtype << endl;
    }*/
    hipMalloc((void**)&cudaIndividualGroups,( groupParts.size() * sizeof ( GroupPart ) ));
    hipMemcpy(cudaIndividualGroups, hostIndividualGroups, ( groupParts.size() * sizeof ( GroupPart ) ), hipMemcpyHostToDevice);

    /// Matrix for RDF calculation
    /// carefull about reactions! !!!!!!!!!!!!!!!!!!!!!!!!!!
    if(simulation->RDFrequired>0){
        hostRDFMatrix = new int[simulation->particleTypes.size()*simulation->particleTypes.size()*simulation->numberOfRDFBins];
        hipMalloc( (void**)&cudaRDFMatrix,  ( simulation->particleTypes.size() * simulation->particleTypes.size() * simulation->numberOfRDFBins *  sizeof ( int ) ));
        hipMemset( cudaRDFMatrix,(int)0, ( simulation->particleTypes.size() * simulation->particleTypes.size() * simulation->numberOfRDFBins * sizeof ( int ) ));
    }

    hipError_t error = hipGetLastError();
    if ( hipSuccess != error ){
        printf( "cuda error during initialization: %s\n",hipGetErrorString(error) );
        return 1;
    }

if(simulation->testmode)
    cout << "initialization done" << endl;

    return 0;
}

int CudaSimulation::copyRDFMatrix(){

    /// copy from GPU
    hipMemcpy(hostRDFMatrix, cudaRDFMatrix, simulation->particleTypes.size() * simulation->particleTypes.size() * simulation->numberOfRDFBins * sizeof ( int ), hipMemcpyDeviceToHost);

    if(simulation->testmode){
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "cuda error: %s\n",hipGetErrorString(error) );
            return 1;
        }
    }

    /// copy to simulation and normalize
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=0; j <simulation->particleTypes.size(); ++j){
            for(int k=0; k<simulation->numberOfRDFBins; ++k){
                simulation->RDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] += (double)hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k]/(double)simulation->numberOfParticlesPerType[i]/(double)simulation->numberOfParticlesPerType[j];
            }
        }
    }

    return 0;
}
int CudaSimulation::copyRDFMatrixToSimulation(){
    return 0;
}

int CudaSimulation::normalizeRDFFRame(){
    /*for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=0; j <simulation->particleTypes.size(); ++j){
            for(int k=0; k<simulation->numberOfRDFBins; ++k){
                /// normalize over particle numbers
                cout << i << "x" << j << "(" << k << "): "<< hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] << " / " <<  simulation->numberOfParticlesPerType[j] << " / " <<  simulation->numberOfParticlesPerType[i] << endl;
                cout << "->" << (double)hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k]/(double)simulation->numberOfParticlesPerType[i]/(double)simulation->numberOfParticlesPerType[j] << endl;
                //hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] = hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k]/simulation->numberOfParticlesPerType[i]/simulation->numberOfParticlesPerType[j];
                /// normalize 2D
                //hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] = hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k];
                /// normalize 3D
                //hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] = hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k];
            }
        }
    }*/
    return 0;
}

int CudaSimulation::callRDFCalculation(){

    //cout << "RDF" << endl;
    calculateRDF<<<gridSize,blockSize>>>(cudaRDFMatrix, cudaCoords, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, simulation->numberParticles, maxCutoff, simulation->particleTypes.size(), simulation->numberOfRDFBins);

    if(simulation->testmode){
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "RDF, cuda error: %s\n",hipGetErrorString(error ));
            return 1;
        }
    }
    ++simulation->numberOfRDFFrames;
    return 0;
}

int CudaSimulation::createNeighborList(){

    numberOfLatticeFields = (boxSize[1]-boxSize[0])/maxCutoff*(boxSize[3]-boxSize[2])/maxCutoff*(boxSize[5]-boxSize[4])/maxCutoff;
    latticeSize = new int[3];
    latticeSize[0] = (boxSize[1]-boxSize[0])/maxCutoff;
    latticeSize[1] = (boxSize[3]-boxSize[2])/maxCutoff;
    latticeSize[2] = (boxSize[5]-boxSize[4])/maxCutoff;
    hipMalloc((void**)&cudaNeighborList,( simulation->numberParticles * 2 * sizeof ( int ) ));
    hipMalloc((void**)&cudaNeighborListBegins,( numberOfLatticeFields * sizeof ( int ) ));

    hostNeighborList = new int[simulation->numberParticles * 2];
    hostNeighborListBegins= new int[numberOfLatticeFields];

    for(int i=0; i<numberOfLatticeFields; ++i){
        hostNeighborListBegins[i]=-1;
    }
    /*if(simulation->testmode){
        cout << "lattice informations:  " << endl;
        cout << "simulation size x[nm]: " << boxSize[1]-boxSize[0] << endl;
        cout << "simulation size y[nm]: " << boxSize[3]-boxSize[2] << endl;
        cout << "simulation size z[nm]: " << boxSize[5]-boxSize[4] << endl;
        cout << "number of voxels:      " << numberOfLatticeFields << endl;
        cout << "voxel edge length:     " << maxCutoff << endl;
        cout << "lattice size x:        " << latticeSize[0] << endl;
        cout << "lattice size y:        " << latticeSize[1] << endl;
        cout << "lattice size z:        " << latticeSize[2] << endl << endl;
    }*/

    for(int i=0; i<simulation->numberParticles; ++i){

        int field=((int)floor((simulation->coords[3*i+2]-boxSize[4])/maxCutoff)%latticeSize[2])*latticeSize[0]*latticeSize[1]
                 +((int)floor((simulation->coords[3*i+1]-boxSize[2])/maxCutoff)%latticeSize[1])*latticeSize[0]
                 +((int)floor((simulation->coords[3*i+0]-boxSize[0])/maxCutoff)%latticeSize[0]);

        if(field<0 || field>numberOfLatticeFields){
            cout << "particle is out of the Box: " << i << " [" <<simulation->coords[3*i+0] << ", " << simulation->coords[3*i+1] << ", " << simulation->coords[3*i+2] << "]" << endl;
            return 1;
        }

        if(hostNeighborListBegins[field]==-1){
            /// this particle is the first in this field. it is its own predecessor and successor
            hostNeighborListBegins[field]=i;
            hostNeighborList[2*i+1]=i;
            hostNeighborList[2*i]=i;
        }
        else{
            /// x f y -> x p f y
            /// particles successor is the fields first particle
            /// S'(p) = f
            hostNeighborList[2*i+1]=hostNeighborListBegins[field];
            /// sucessor of the first particles predecessor is the particle
            /// S(P(f))=p , P(f)=x -> S'(x)=p
            hostNeighborList[2*hostNeighborList[2*hostNeighborListBegins[field]]+1]=i;
            /// particles predecessor is the predecessor of the fields first particle
            /// P'(p)=P(f)=x
            hostNeighborList[2*i]=hostNeighborList[2*hostNeighborListBegins[field]];
            /// fields first particles new predecessor is the current particle
            /// P'(f)=p
            hostNeighborList[2*hostNeighborListBegins[field]]=i;
            //hostNeighborListBegins[field]=i;
        }
    }

    hipError_t error = hipGetLastError();
    if ( hipSuccess != error ){
        printf( "cuda error: %s\n",hipGetErrorString(error) );
        return 1;
    }

    return 0;
}

int CudaSimulation::testNeighborList(){

    hipMemcpy(hostNeighborList, cudaNeighborList, ( simulation->numberParticles * 2 * sizeof ( int ) ), hipMemcpyDeviceToHost);
    hipMemcpy(hostNeighborListBegins, cudaNeighborListBegins, ( numberOfLatticeFields * sizeof ( int ) ), hipMemcpyDeviceToHost);

    int count = 0;
    int count2 = 0;
    int x;
    for(int i=0; i<numberOfLatticeFields; ++i){
        x=hostNeighborListBegins[i];
        //cout << i << ":" << x << endl;
        if(x!=-1){
            do{
                count++;
                //cout << hostNeighborList[2*x+0] << " " << x << " " << hostNeighborList[2*x+1] << endl;
                x=hostNeighborList[2*x+1];
                if(x==hostNeighborListBegins[i])
                    break;
                //char a;
                //cin >> a;
            }while(true);
        }
        else{
            ++count2;
        }
    }
    cout << "Neighborlist check:" <<  "count: "<< count << "    part num: " << simulation->numberParticles << "     (check 2:" << count2 << " empty fields)"<<  endl;
    if(count!=simulation->numberParticles){
        cout << "Neighborlist broken!" << endl;
        return 1;
    }

    //cout << "Neighborlist okay!" << endl;

    hipError_t error = hipGetLastError();
    if ( hipSuccess != error ){
        printf( "cuda error: %s\n",hipGetErrorString(error) );
        return 1;
    }

    return 0;
}

int CudaSimulation::copyPosToDevice(){

    hipMemcpy(cudaCoords, simulation->coords, simulation->numberParticles * 3 * sizeof(double), hipMemcpyHostToDevice);

    if(simulation->testmode){
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "cuda error: %s\n",hipGetErrorString(error) );
            return 1;
        }
    }
    return 0;
}

int CudaSimulation::copyPosFromDevice(){

    hipMemcpy(simulation->coords, cudaCoords, simulation->numberParticles * 3 * sizeof ( double ), hipMemcpyDeviceToHost);

    if(simulation->testmode){
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "cuda error: %s\n",hipGetErrorString(error) );
            return 1;
        }
    }
    return 0;
}


int CudaSimulation::simulate(){


    //cout << "order one" << endl;
    //orderOne<<<1,1>>>(cudaCoords, cudaForces, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, globalRandStates, simulation->numberParticles, maxCutoff, cudaOrderOnePotentialsMatrix, cudaCudaOrderOnePotentials, simulation->orderOnePotentials.size(), simulation->particleTypes.size(), cudaCollisionRadiiMatrix);
    orderOne<<<gridSize,blockSize>>>(cudaCoords, cudaForces, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, globalRandStates, simulation->numberParticles, maxCutoff, cudaOrderOnePotentialsMatrix, cudaCudaOrderOnePotentials, simulation->orderOnePotentials.size(), simulation->particleTypes.size(), cudaParticleRadiiMatrix);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "order one, cuda error: %s\n",hipGetErrorString(error) );
                return 1;
            }
    }
    //cout << "order two" << endl;
    /*orderTwo<<<gridSize,blockSize>>>(cudaCoords, cudaForces, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, globalRandStates, simulation->numberParticles, maxCutoff, cudaOrderTwoPotentialsMatrix, cudaCudaOrderTwoPotentials, simulation->orderTwoPotentials.size(), simulation->particleTypes.size(), cudaCollisionRadiiMatrix);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "order two, cuda error: %s\n",hipGetErrorString(error) );
                return 1;
            }
    }*/
    //cout << "order two" << endl;
    int warpsize=32;
    warpOrderTwo<<<numberOfLatticeFields,warpsize, (warpsize*3*4*sizeof(double)+(27+warpsize*3+5)*sizeof(int))>>>(warpsize, cudaCoords, cudaForces, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, globalRandStates, simulation->numberParticles, maxCutoff, cudaOrderTwoPotentialsMatrix, cudaCudaOrderTwoPotentials, simulation->orderTwoPotentials.size(), simulation->particleTypes.size(), cudaCollisionRadiiMatrix);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "warp order two, cuda error: %s\n",hipGetErrorString(error) );
                return 1;
            }
    }
    //cout << "groups" << endl;
    /// TODO!: use different grid and block size!
        groups<<<gridSize,blockSize>>>(cudaCoords, cudaForces, cudaTypes, simulation->numberParticles, simulation->particleTypes.size(), cudaCollisionRadiiMatrix, cudaCudaGroupPotentials, cudaIndividualGroups, numberOfIndividualGroups, cudaBoxSize);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "groups, cuda error: %s\n",hipGetErrorString(error) );
                return 1;
            }
    }

    //cout << "update" << endl;
    update<<<gridSize,blockSize>>>(  cudaCoords, cudaForces, cudaTypes, cudaD, cudaNeighborList, cudaNeighborListBegins, cudaBoxSize, cudaSemaphore, globalRandStates, simulation->stepSizeInPs, simulation->numberParticles, simulation->boltzmann,  simulation->temperature, maxCutoff, cudaLatticeSize);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "update, cuda error: %s\n",hipGetErrorString(error ));
                return 1;
            }
    }

    return 0;
}


/// /////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// cuda kernels ////////////////////////////////////////////////////////////////////////////////////////////////
/// /////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void orderOne(double* cudaCoords, double* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, double * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderOnePotentialsMatrix, CudaOrderOnePotential * cudaCudaOrderOnePotentials, int numberOfOrderOnePotentials, int numberOfParticleTypes, double * cudaParticleRadiiMatrix){


    int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;

    ////////////////////////////////////////////////////
    bool considerParticleRadius = true;
    //////////////////////////////////


    if(particleNumber<numberParticles){
        hiprandState localState = globalRandStates[particleNumber];

        /// do calculation of forces and maybe reactions here ...

        /// go through all order one potetntials
        for(int orderOnePotential=0; orderOnePotential<numberOfOrderOnePotentials; ++orderOnePotential){
            ///  lookup in matrix whether they apply to the current particle type
            if(cudaOrderOnePotentialsMatrix[orderOnePotential*numberOfParticleTypes+cudaTypes[particleNumber]]==1){
                /// check what kind of potential it is
                if(cudaCudaOrderOnePotentials[orderOnePotential].type==1){/// Disk

                    /// calculation depends on the normal vector. assign x,y and z coordinates to variables
                    int normal, side1, side2;
                    /// normal vector on x axis -> assign x to normal and y and z to the lateral (on Disk) directions
                    if(cudaCudaOrderOnePotentials[orderOnePotential].normal[0]==1){
                        normal=0;side1=1;side2=2;
                    }
                    /// y
                    else if(cudaCudaOrderOnePotentials[orderOnePotential].normal[1]==1){
                        normal=1;side1=0;side2=2;
                    }
                    /// x
                    else {
                        normal=2;side1=1;side2=0;
                    }

                    /// different subtypes
                    if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==1){/// attractive

                        double r = cudaCudaOrderOnePotentials[orderOnePotential].origin[normal]-cudaCoords[3*particleNumber+normal];
                        cudaForces[3*particleNumber+normal]+=-cudaCudaOrderOnePotentials[orderOnePotential].forceConst*r;

                        /// particle radius!
                        r = sqrt(
                                    pow(cudaCoords[3*particleNumber+side1]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side1],2)
                                    +
                                    pow(cudaCoords[3*particleNumber+side2]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side2],2)
                                );
                        if (r > cudaCudaOrderOnePotentials[orderOnePotential].radius) {
                            cudaForces[3*particleNumber+side1]+=
                                    -cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                    *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                    /r
                                    *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side1]-cudaCoords[3*particleNumber+side1]);
                            cudaForces[3*particleNumber+side2]+=
                                    -cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                    *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                    /r
                                    *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side2]-cudaCoords[3*particleNumber+side2]);

                        }
                    }
                    else if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==2){/// repulsive
                        // makes no sense ...
                        /*
                        // force along normal vector
                        r = distToDiskPlane;// actual
                        r0 = pRadius;// desired
                        double r_1 = distToCenterWithinDiskPlane - pRadius;
                        double r0_1 = diskRadius;
                        if (r < r0 && r_1 < r0_1) {

                            precompute = (k * (-r0 + r) / r);

                                gradient[0] = gradient[0]+ precompute * ( pointOnDiskPlane[0]-coords1[0]);
                                gradient[1] = gradient[1]+ precompute * ( pointOnDiskPlane[1]-coords1[1]);
                                gradient[2] = gradient[2]+ precompute * ( pointOnDiskPlane[2]-coords1[2]);

                        }*/
                    }
                }/// end Disk
                else if(cudaCudaOrderOnePotentials[orderOnePotential].type==2){/// Cylinder

                        int normal, side1, side2;
                        if(cudaCudaOrderOnePotentials[orderOnePotential].normal[0]==1){normal=0;side1=1;side2=2;}
                        else if(cudaCudaOrderOnePotentials[orderOnePotential].normal[1]==1){normal=1;side1=0;side2=2;}
                        else {normal=2;side1=1;side2=0;}

                        if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==1){/// attractive

                            double r = cudaCudaOrderOnePotentials[orderOnePotential].origin[normal]-cudaCoords[3*particleNumber+normal];
                            if(fabsf(r)>cudaCudaOrderOnePotentials[orderOnePotential].height*0.5)
                            cudaForces[3*particleNumber+normal]+=-cudaCudaOrderOnePotentials[orderOnePotential].forceConst*(fabsf(r)-cudaCudaOrderOnePotentials[orderOnePotential].height*0.5)/fabsf(r)*r;

                            /// particle radius!
                            r = sqrt(
                                        pow(cudaCoords[3*particleNumber+side1]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side1],2)
                                        +
                                        pow(cudaCoords[3*particleNumber+side2]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side2],2)
                                    );
                            if (r > cudaCudaOrderOnePotentials[orderOnePotential].radius) {
                                cudaForces[3*particleNumber+side1]+=
                                        -cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                        *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                        /r
                                        *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side1]-cudaCoords[3*particleNumber+side1]);
                                cudaForces[3*particleNumber+side2]+=
                                        -cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                        *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                        /r
                                        *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side2]-cudaCoords[3*particleNumber+side2]);

                            }
                        }
                        else if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==2){/// repulsive

                            double r = cudaCudaOrderOnePotentials[orderOnePotential].origin[normal]-cudaCoords[3*particleNumber+normal];
                            if(fabsf(r)<cudaCudaOrderOnePotentials[orderOnePotential].height*0.5)
                            cudaForces[3*particleNumber+normal]+=cudaCudaOrderOnePotentials[orderOnePotential].forceConst*(fabsf(r)-cudaCudaOrderOnePotentials[orderOnePotential].height*0.5)/fabsf(r)*r;

                            /// particle radius!
                            r = sqrt(
                                        pow(cudaCoords[3*particleNumber+side1]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side1],2)
                                        +
                                        pow(cudaCoords[3*particleNumber+side2]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side2],2)
                                    );
                            if (r < cudaCudaOrderOnePotentials[orderOnePotential].radius) {
                                cudaForces[3*particleNumber+side1]+=
                                        cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                        *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                        /r
                                        *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side1]-cudaCoords[3*particleNumber+side1]);
                                cudaForces[3*particleNumber+side2]+=
                                        cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                        *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                        /r
                                        *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side2]-cudaCoords[3*particleNumber+side2]);

                            }
                        }
                }/// end Cylinder
                else if(cudaCudaOrderOnePotentials[orderOnePotential].type==3){/// Sphere
                        double dist = 0;
                        for(int dim=0; dim<3; ++dim){
                            dist += (cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]-cudaCoords[3*particleNumber+dim])*(cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]-cudaCoords[3*particleNumber+dim]);
                        }
                        dist = sqrt(dist);

                        if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==1 && dist>cudaCudaOrderOnePotentials[orderOnePotential].radius){/// attractive
                            double precompute = cudaCudaOrderOnePotentials[orderOnePotential].forceConst * (dist - cudaCudaOrderOnePotentials[orderOnePotential].radius) / dist;
                            for(int dim=0; dim<3; ++dim){
                                cudaForces[3*particleNumber+dim] += -precompute * (cudaCudaOrderOnePotentials[orderOnePotential].origin[dim] - cudaCoords[3*particleNumber+dim]);
                            }
                        }
                        else if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==2 && dist<cudaCudaOrderOnePotentials[orderOnePotential].radius){/// repulsive
                            double precompute = cudaCudaOrderOnePotentials[orderOnePotential].forceConst * (dist - cudaCudaOrderOnePotentials[orderOnePotential].radius) / dist;
                            for(int dim=0; dim<3; ++dim){
                                cudaForces[3*particleNumber+dim] += -precompute * (cudaCudaOrderOnePotentials[orderOnePotential].origin[dim] - cudaCoords[3*particleNumber+dim]);
                            }
                        }
                }/// end Sphere
                else if(cudaCudaOrderOnePotentials[orderOnePotential].type==4){/// Box
                        if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==1){/// attractive
                            for(int dim=0; dim<3; ++dim){
                                double distToBoxBegin = cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]+cudaCudaOrderOnePotentials[orderOnePotential].extension[dim]-(considerParticleRadius ? cudaParticleRadiiMatrix[cudaTypes[particleNumber]] : 0);
                                if(cudaCoords[3*particleNumber+dim] > distToBoxBegin){
                                    cudaForces[3*particleNumber+dim]+= -cudaCudaOrderOnePotentials[orderOnePotential].forceConst *
                                            (distToBoxBegin - cudaCoords[3*particleNumber+dim]);
                                }
                                else{
                                    double distToBoxEnd = cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]+(considerParticleRadius ? cudaParticleRadiiMatrix[cudaTypes[particleNumber]] : 0);
                                    if(cudaCoords[3*particleNumber+dim] < distToBoxEnd){
                                    cudaForces[3*particleNumber+dim]+= cudaCudaOrderOnePotentials[orderOnePotential].forceConst *
                                            (cudaCoords[3*particleNumber+dim]-distToBoxEnd);
                                    }
                                }
                            }
                        }
                        else if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==2){/// repulsive
                            for(int dim=0; dim<3; ++dim){
                                double distToBoxBegin = cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]+cudaCudaOrderOnePotentials[orderOnePotential].extension[dim]+(considerParticleRadius ? cudaParticleRadiiMatrix[cudaTypes[particleNumber]] : 0);
                                if(cudaCoords[3*particleNumber+dim] > distToBoxBegin){
                                    cudaForces[3*particleNumber+dim] += cudaCudaOrderOnePotentials[orderOnePotential].forceConst *
                                            (distToBoxBegin - cudaCoords[3*particleNumber+dim]);
                                }
                                else{
                                    double distToBoxEnd = cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]-(considerParticleRadius ? cudaParticleRadiiMatrix[cudaTypes[particleNumber]] : 0);
                                    if(cudaCoords[3*particleNumber+dim] < distToBoxEnd){
                                    cudaForces[3*particleNumber+dim] += -cudaCudaOrderOnePotentials[orderOnePotential].forceConst *
                                            (cudaCoords[3*particleNumber+dim]-distToBoxEnd);
                                    }
                                }
                            }
                        }
                }/// end Box
            }/// endif order one potentials matrix
        }/// end iterate over order one potentials

        globalRandStates[particleNumber] = localState;
    }
    return;
}

__device__ void calculateOrderTwoPotential(int particleNumber, int interactingParticle, int orderTwoPotentialNr, double r, double* cudaCoords, double* cudaForces, int* cudaTypes, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix){

    double r0 = cudaCollisionRadiiMatrix[cudaTypes[particleNumber]*numberOfParticleTypes+cudaTypes[interactingParticle]];
    if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].type==1){        /// Harmonic Potential
        double precompute = cudaCudaOrderTwoPotentials[orderTwoPotentialNr].forceConst * (r-r0)/r;
        for(int coord=0; coord<3; ++coord){
            double force = precompute * (cudaCoords[interactingParticle*3+coord]-cudaCoords[particleNumber*3+coord]);
            if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==1 && r>r0){ /// attractive
                atomicAdd(&cudaForces[particleNumber*3+coord], -force );
                atomicAdd(&cudaForces[interactingParticle*3+coord], force );
            }
            else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==2 && r<r0){ /// repulsive
                atomicAdd(&cudaForces[particleNumber*3+coord], -force );
                atomicAdd(&cudaForces[interactingParticle*3+coord], force );
            }
            else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==3){ /// spring
                atomicAdd(&cudaForces[particleNumber*3+coord], -force );
                atomicAdd(&cudaForces[interactingParticle*3+coord], force );
            }
        }
    }
    else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].type==2){     /// Harmonic weak interaction Potential
        double iradius = cudaCudaOrderTwoPotentials[orderTwoPotentialNr].length + r0;
        if (r < iradius && r > r0) {
            double precompute =  ( cudaCudaOrderTwoPotentials[orderTwoPotentialNr].forceConst * (-r0 + r) * (iradius -r));
            for(int coord=0; coord<3; ++coord){
                double force = precompute * (cudaCoords[interactingParticle*3+coord]-cudaCoords[particleNumber*3+coord]);
                atomicAdd(&cudaForces[particleNumber*3+coord], -force );
                atomicAdd(&cudaForces[interactingParticle*3+coord], force );
            }
        }
    }
}

__device__ void getNeighbors(int particleNumber, int * todo, double* cudaCoords, int * cudaLatticeSize, double * cudaBoxSize, int maxCutoff){

    int x,y,z;
    int field=((int)floor((cudaCoords[3*particleNumber+2]-cudaBoxSize[4])/maxCutoff)%cudaLatticeSize[2])*cudaLatticeSize[0]*cudaLatticeSize[1]
             +((int)floor((cudaCoords[3*particleNumber+1]-cudaBoxSize[2])/maxCutoff)%cudaLatticeSize[1])*cudaLatticeSize[0]
             +((int)floor((cudaCoords[3*particleNumber+0]-cudaBoxSize[0])/maxCutoff)%cudaLatticeSize[0]);

    /// surrounding, for calculation imprtant fields
    /// TODO: CHECK! !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

        for(x=-1; x<2;x++){
            for(y=-1; y<2;y++){
                for(z=-1; z<2;z++){
                    todo[(x+1)+(y+1)*3+(z+1)*9]=
                    (
                        (
                            (field%(cudaLatticeSize[0]))
                            +x+cudaLatticeSize[0]
                        )
                        %cudaLatticeSize[0]
                    )

                    +cudaLatticeSize[0]*
                    (
                        (
                            (int)floorf
                            (
                                (float)(field%(cudaLatticeSize[0]*cudaLatticeSize[1]))
                                /
                                (float)(cudaLatticeSize[0])
                            )
                            +y+cudaLatticeSize[1]
                        )
                        %cudaLatticeSize[1]
                    )

                    +cudaLatticeSize[0]*cudaLatticeSize[1]*
                    (
                        (
                            (int)floorf
                            (
                                (float)(field)
                                /
                                (float)(cudaLatticeSize[0]*cudaLatticeSize[1])
                            )
                            +z+cudaLatticeSize[2]
                        )
                        %cudaLatticeSize[2]
                    );
                }
            }
        }


}

__global__ void orderTwo(double* cudaCoords, double* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, double * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderTwoPotentialsMatrix, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfOrderTwoPotentials, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix){

    int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;

    if(particleNumber<numberParticles){
        //hiprandState localState = globalRandStates[particleNumber];

        int todo[27];

        getNeighbors(particleNumber, todo, cudaCoords, cudaLatticeSize, cudaBoxSize, maxCutoff);

        /// do calculation of forces and maybe reactions here ...

            /** TODO!!:
             *bring matrices to shared mem
             *interaction matrix could contain the specific cutoffs
             *think about parameter storage
             *then dynamik arrays
             *later think about accellerations due to ideas below
             *
             * need:   - interaction matices (radii, forces) for every force (aligned in one array, +array size)
             *      - more parameter? how to store?
             *  data alignment: x,y,z,type,rand?,force?
             *
             * call voxel per warp(n threads)
             * load first n coords in shared mem
             * calculate distances to particles in surrounding fields (always load one particle and calc. n distances)
             *
             * check for periodic boundaries ...
             */


            /// loop over all "todo" fields around the current field
            for(int x=0; x<27; x++){
                /// begin link to the first element from the list of the field
                int interactingParticle=cudaNeighborListBegins[todo[x]];
                if(interactingParticle!=-1){
                    do
                    {
                        //if(interactingParticle!=particleNumber){
                        /// calculating interaction just once, and apply it for both particles
                        if(interactingParticle<particleNumber){

                            float r=0.0f;
                            float rij[3];
                            for (int dim=0;dim<3;dim++){
                                rij[dim]=cudaCoords[3*particleNumber+dim]-cudaCoords[3*interactingParticle+dim];
                                if(rij[dim]>( (cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
                                if(rij[dim]<(-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]+(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
                                r=r+(rij[dim]*rij[dim]);
                            }
                            r=sqrtf(r);

                            for(int orderTwoPotentialNr=0; orderTwoPotentialNr<numberOfOrderTwoPotentials; ++orderTwoPotentialNr){
                                if(cudaOrderTwoPotentialsMatrix[cudaTypes[particleNumber]*numberOfParticleTypes+cudaTypes[interactingParticle]*numberOfParticleTypes+orderTwoPotentialNr]!=0 && r<=maxCutoff){
                                    calculateOrderTwoPotential(particleNumber, interactingParticle, orderTwoPotentialNr, r, cudaCoords, cudaForces, cudaTypes, cudaCudaOrderTwoPotentials, numberOfParticleTypes, cudaCollisionRadiiMatrix);
                                }
                            }
                        }
                        interactingParticle=cudaNeighborList[2*interactingParticle+1];
                    }
                    while(interactingParticle!=cudaNeighborListBegins[todo[x]]);/// loop/list end
                }
            }
        //globalRandStates[particleNumber] = localState;
    }
    return;
}

__device__ void warpGetNeighbors(int field, int * todo, int * cudaLatticeSize){

    if(threadIdx.x<27){
        int x,y,z;

        /// /////////////////////////////////////////////////////////////
        /// modulo is slow!!
        /// /////////////////////////////////////////////////////////////

        x=threadIdx.x/9%3 -1;
        y=threadIdx.x/3%3 -1;
        z=threadIdx.x%3 -1;

        //for(x=-1; x<2;x++){
            //for(y=-1; y<2;y++){
                //for(z=-1; z<2;z++){
                    todo[(x+1)+(y+1)*3+(z+1)*9]=
                    (
                        (
                            (field%(cudaLatticeSize[0]))
                            +x+cudaLatticeSize[0]
                        )
                        %cudaLatticeSize[0]
                    )

                    +cudaLatticeSize[0]*
                    (
                        (
                            (int)floorf
                            (
                                (float)(field%(cudaLatticeSize[0]*cudaLatticeSize[1]))
                                /
                                (float)(cudaLatticeSize[0])
                            )
                            +y+cudaLatticeSize[1]
                        )
                        %cudaLatticeSize[1]
                    )

                    +cudaLatticeSize[0]*cudaLatticeSize[1]*
                    (
                        (
                            (int)floorf
                            (
                                (float)(field)
                                /
                                (float)(cudaLatticeSize[0]*cudaLatticeSize[1])
                            )
                            +z+cudaLatticeSize[2]
                        )
                        %cudaLatticeSize[2]
                    );
                //}
            //}
        //}
    }
}

__device__ void calculateOrderTwoPotential(double * particleCoord, double * particleForce, int particleType, double * interactingParticleCoord, double * interactingParticleForce, int interactingParticleType, int orderTwoPotentialNr, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, double * cudaBoxSize, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix){

    float r=0.0f;
    float rij[3];
    for (int dim=0;dim<3;dim++){
        rij[dim]=particleCoord[dim]-interactingParticleCoord[dim];
        if(rij[dim]>( (cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
        if(rij[dim]<(-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]+(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
        r=r+(rij[dim]*rij[dim]);
    }
    r=sqrtf(r);

    double r0 = cudaCollisionRadiiMatrix[particleType*numberOfParticleTypes+interactingParticleType];
    if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].type==1){        /// Harmonic Potential
        double precompute = cudaCudaOrderTwoPotentials[orderTwoPotentialNr].forceConst * (r-r0)/r;
        for(int dim=0; dim<3; ++dim){
        /*    double force = precompute * (interactingParticleCoord[dim]-particleCoord[dim]);
            if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==1 && r>r0){ /// attractive
                atomicAdd(&particleForce[dim], -force );
                atomicAdd(&interactingParticleForce[dim], force );
            }
            else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==2 && r<r0){ /// repulsive
                atomicAdd(&particleForce[dim], -force );
                atomicAdd(&interactingParticleForce[dim], force );
            }
            else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==3){ /// spring
                atomicAdd(&particleForce[dim], -force );
                atomicAdd(&interactingParticleForce[dim], force );
            }*/
        }
    }
    else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].type==2){     /// Harmonic weak interaction Potential
        double iradius = cudaCudaOrderTwoPotentials[orderTwoPotentialNr].length + r0;
        if (r < iradius && r > r0) {
            double precompute =  ( cudaCudaOrderTwoPotentials[orderTwoPotentialNr].forceConst * (-r0 + r) * (iradius -r));
            for(int dim=0; dim<3; ++dim){
                double force = precompute * (interactingParticleCoord[dim]-particleCoord[dim]);
              //  atomicAdd(&particleForce[dim], -force );
              //  atomicAdd(&interactingParticleForce[dim], force );
            }
        }
    }
}

__global__ void warpOrderTwo(int warpsize, double* cudaCoords, double* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, double * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderTwoPotentialsMatrix, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfOrderTwoPotentials, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix){

    //int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;
    //hiprandState localState;
    //hiprandState localState = globalRandStates[particleNumber]
    ;
    int thread = threadIdx.x;
    int field = blockIdx.x;
    int particleNumber = cudaNeighborListBegins[field];
    if(particleNumber==-1){
        return;
    }

    /// link to shared memory
    extern __shared__ double s[];
    //int *integerData = s;                        // nI ints
    //float *floatData = (float*)&integerData[nI]; // nF floats
    //char *charData = (char*)&floatData[nF];      // nC chars

    //__shared__ double localParticleCoords[warpsize*3];
    double * localParticleCoords = s;
    //__shared__ double localParticleForces[warpsize*3];
    double * localParticleForces = &localParticleCoords[warpsize*3];
    //__shared__ double localInteractingParticleCoords[warpsize*3];
    double * localInteractingParticleCoords = &localParticleForces[warpsize*3];
    //__shared__ double localInteractingParticleForces[warpsize*3];
    double * localInteractingParticleForces = &localInteractingParticleCoords[warpsize*3];

    //__shared__ int todo[27];
    int * todo = (int*)&localInteractingParticleForces[warpsize*3];
    //__shared__ int localParticleTypes[warpsize];
    int * localParticleTypes = &todo[27];
    //__shared__ int localInteractingParticleTypes[warpsize];
    int * localInteractingParticleTypes = &localParticleTypes[warpsize];
    //__shared__ int localInteractingParticleNumber[warpsize];
    int * localInteractingParticleNumber = &localInteractingParticleTypes[warpsize];
    //__shared__ int lastParticle;
    int * lastParticle = &localInteractingParticleNumber[warpsize];
    //__shared__ int end;
    int * end = &lastParticle[1];
    //__shared__ int todoEnd;
    int * todoEnd = &end[1];
    //__shared__ int lastTodoField;
    int * lastTodoField = &todoEnd[1];
    //__shared__ int lastInteractingParticle;
    int * lastInteractingParticle = &lastTodoField[1];
    /// sum = (warpsize*3*4*sizeof(double)+(27+warpsize*3+5)*sizeof(int))

    warpGetNeighbors(field, todo, cudaLatticeSize);

    /// iterate over particles in this field
    do{
        /// assign each thread a particle //////////////////////////////////////////////////////////////////
        for(int i=0; i<thread; i++){
            particleNumber = cudaNeighborList[particleNumber*2+1];
            if(particleNumber == cudaNeighborListBegins[field]){
                atomicExch(end, 1);
                return;
            }
        }
        /// store particle properties local
        for(int dim=0; dim<3; ++dim){
            localParticleCoords[3*thread+dim]=cudaCoords[3*particleNumber+dim];
            localParticleForces[3*thread+dim]=0;
        }
        localParticleTypes[thread]=cudaTypes[particleNumber];


        int x=0;
        int interactingParticleNumber=cudaNeighborListBegins[todo[x]];

        /// for all interacting fields ///////////////////////////////////////////////////////////////////
        do{
            /// get one interacting particle for all threads
            for(int i=0; i<=thread && x>27; i++){
                /// first(0) thread gets first particle in list
                /// if it occures again, we had all from this field
                if(interactingParticleNumber == cudaNeighborListBegins[todo[x]] && i!=0){
                    ++x;    /// next todo field
                    interactingParticleNumber = cudaNeighborListBegins[todo[x]];   /// start at next todo field
                    --i;
                    continue;
                }
                /// field is empty
                if(interactingParticleNumber==-1){
                    ++x;    /// next todo field
                    interactingParticleNumber = cudaNeighborListBegins[todo[x]];   /// start at next todo field
                    --i;
                    continue;
                }
                interactingParticleNumber = cudaNeighborList[interactingParticleNumber*2+1];
            }
            /// store interacting particle properties local
            for(int dim=0; dim<3; ++dim){
                localInteractingParticleCoords[3*thread+dim]=cudaCoords[3*interactingParticleNumber+dim];
                localInteractingParticleForces[3*thread+dim]=0;
            }
            localInteractingParticleTypes[thread]=cudaTypes[interactingParticleNumber];
            localInteractingParticleNumber[thread]=interactingParticleNumber;



            /// calc
            /// iterate more intelligent -> not double, order
            for(int interactingThread=0; interactingThread<warpsize; ++interactingParticleNumber){
                interactingParticleNumber = localInteractingParticleNumber[interactingThread];
                if(interactingParticleNumber<particleNumber){

                    for(int orderTwoPotentialNr=0; orderTwoPotentialNr<numberOfOrderTwoPotentials; ++orderTwoPotentialNr){
                        if(cudaOrderTwoPotentialsMatrix[cudaTypes[particleNumber]*numberOfParticleTypes+cudaTypes[interactingParticleNumber]*numberOfParticleTypes+orderTwoPotentialNr]!=0){
                            //__device__ void calculateOrderTwoPotential(double * particleCoord, double * particleForce, int particleType, double * interactingParticleCoord, double * interactingParticleForce, int interactingParticleType, int orderTwoPotentialNr, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, double * cudaBoxSize, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix);
                            calculateOrderTwoPotential(&localParticleCoords[thread], &localParticleForces[thread], localParticleTypes[thread], &localInteractingParticleCoords[interactingParticleNumber], &localInteractingParticleForces[interactingParticleNumber], localInteractingParticleTypes[interactingParticleNumber], orderTwoPotentialNr, cudaCudaOrderTwoPotentials, cudaBoxSize, numberOfParticleTypes, cudaCollisionRadiiMatrix);
                            //calculateOrderTwoPotential(particleNumber, interactingParticle, orderTwoPotentialNr, r, cudaCoords, cudaForces, cudaTypes, cudaCudaOrderTwoPotentials, numberOfParticleTypes, cudaCollisionRadiiMatrix);
                        }
                    }
                }
            }

            interactingParticleNumber = localInteractingParticleNumber[thread];
            /// just the last thread, with the last interacting particle stores its number and field, so that the threads can start again from there.
            if(blockIdx.x==warpsize){
                /// there is a next particle in this field
                if(cudaNeighborList[interactingParticleNumber*2+1]!=cudaNeighborListBegins[x]){
                    /// store the particle and field number
                    atomicExch(lastInteractingParticle, cudaNeighborList[interactingParticleNumber*2+1]);
                    atomicExch(lastTodoField, x);
                    atomicExch(todoEnd, 0);    /// we are not done jet
                }
                /// if we are done with this todo field
                else{
                    atomicExch(todoEnd, 1);    /// we are done, if there are no more particle -> check
                    /// as long, as we have fields to consider (todo)
                    while(x>27){
                        ++x;    /// take next todo field
                        if(cudaNeighborListBegins[todo[x]]==-1)   /// empty?
                            continue;                       /// again. -> take next
                        else{   /// todo field not empty -> store field and first particle
                            atomicExch(lastInteractingParticle, cudaNeighborListBegins[todo[x]]);
                            atomicExch(lastTodoField, x);
                            atomicExch(todoEnd, 0);    /// we are not done jet
                        }
                    }
                }
            }
            /// block wise sync
            __syncthreads();
            x=lastTodoField[0];
            interactingParticleNumber=lastInteractingParticle[0];
        }while(todoEnd[0]==0); /// end interacting fields

        /// store locally saved changes global (interacting particles)

        /// if more particle in this field continue
        if(blockIdx.x==warpsize){
            if(cudaNeighborList[particleNumber*2+1]!=cudaNeighborListBegins[field]){
                atomicExch(lastParticle, cudaNeighborList[particleNumber*2+1]);
                atomicExch(end, 0);
            }
            else{
                atomicExch(end, 1);
            }
        }
        /// block wise sync
        __syncthreads();
        particleNumber=lastParticle[0];
    }while(end[0]==0); /// end loop over particles from this field

    /// store locally saved changes global (particles)

    //globalRandStates[particleNumber] = localState;
    return;
}

__global__ void calculateRDF(int * cudaRDFMatrix, double* cudaCoords, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, double * cudaBoxSize, int numberParticles, int maxCutoff, int numberOfParticleTypes, int numberOfBins){

    int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;

    if(particleNumber<numberParticles){

        int todo[27];

        getNeighbors(particleNumber, todo, cudaCoords, cudaLatticeSize, cudaBoxSize, maxCutoff);

        /// loop over all "todo" fields around the current field
        for(int x=0; x<27; x++){
            /// begin link to the first element from the list of the field
            int interactingParticle=cudaNeighborListBegins[todo[x]];
            if(interactingParticle!=-1){
                do
                {
                    //if(interactingParticle!=particleNumber){
                    /// calculating interaction just once, and apply it for both particles
                    if(interactingParticle<particleNumber){

                        float r=0.0f;
                        float rij[3];
                        for (int dim=0;dim<3;dim++){
                            rij[dim]=cudaCoords[3*particleNumber+dim]-cudaCoords[3*interactingParticle+dim];
                            if(rij[dim]>((cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
                            if(rij[dim]<(-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]+(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
                            r=r+(rij[dim]*rij[dim]);
                        }
                        r=sqrtf(r);
/// carefull about reactions! !!!!!!!!!!!!!!!!!!!!!!!!!!
                        if(r<maxCutoff){
                            int bin = (int)(r*numberOfBins/maxCutoff);
                            atomicAdd(&cudaRDFMatrix[cudaTypes[particleNumber]*numberOfParticleTypes*numberOfBins+cudaTypes[interactingParticle]*numberOfBins+bin], 1);
                            atomicAdd(&cudaRDFMatrix[cudaTypes[interactingParticle]*numberOfParticleTypes*numberOfBins+cudaTypes[particleNumber]*numberOfBins+bin], 1);
                        }
                    }
                    interactingParticle=cudaNeighborList[2*interactingParticle+1];
                }
                while(interactingParticle!=cudaNeighborListBegins[todo[x]]);/// loop/list end
            }
        }
    }
}

__global__ void groups(double* cudaCoords, double* cudaForces, int* cudaTypes, int numberParticles, int numberOfParticleTypes, double * cudaCollisionRadiiMatrix, CudaOrderTwoPotential * cudaCudaGroupPotentials, GroupPart * cudaIndividualGroups, int numberOfIndividualGroups, double * cudaBoxSize){

    int pairNumber=blockIdx.x * blockDim.x + threadIdx.x;

    if(pairNumber<numberOfIndividualGroups){
        float r=0.0f;
        float rij[3];
        for (int dimension=0;dimension<3;dimension++){
            rij[dimension]=cudaCoords[3*cudaIndividualGroups[pairNumber].particle1+dimension]-cudaCoords[3*cudaIndividualGroups[pairNumber].particle2+dimension];
            if(rij[dimension]>((cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2])/2)){rij[dimension]=rij[dimension]-(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2]);}
            if(rij[dimension]<(-(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2])/2)){rij[dimension]=rij[dimension]+(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2]);}
            r=r+(rij[dimension]*rij[dimension]);
        }
        r=sqrtf(r);

        calculateOrderTwoPotential(cudaIndividualGroups[pairNumber].particle1, cudaIndividualGroups[pairNumber].particle2, cudaIndividualGroups[pairNumber].groupPot, r, cudaCoords, cudaForces, cudaTypes, cudaCudaGroupPotentials, numberOfParticleTypes, cudaCollisionRadiiMatrix);
    }
}

__global__ void update(double* cudaCoords, double* cudaForces, int* cudaTypes, double* cudaD, int * cudaNeighborList, int * cudaNeighborListBegins, double * cudaBoxSize, int * cudaSemaphore, hiprandState* globalRandStates, double dt, int numberParticles, double KB, double T, double maxCutoff, int * cudaLatticeSize){

    int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;

    if(particleNumber<numberParticles){
        hiprandState localState = globalRandStates[particleNumber];

        int oldVoxel=   ((int)floor((cudaCoords[3*particleNumber+2]-cudaBoxSize[4])/maxCutoff)%cudaLatticeSize[2])*cudaLatticeSize[0]*cudaLatticeSize[1]
                        +((int)floor((cudaCoords[3*particleNumber+1]-cudaBoxSize[2])/maxCutoff)%cudaLatticeSize[1])*cudaLatticeSize[0]
                        +((int)floor((cudaCoords[3*particleNumber+0]-cudaBoxSize[0])/maxCutoff)%cudaLatticeSize[0]);

        for(int dimension=0; dimension<3; ++dimension){
            /// apply diffusion and forces -> update positions
            /// x(t+dt) = x(t) - dt*D*(F(x(t))/kT) + sqrt(2Ddt)*N(0,1)
            cudaCoords[particleNumber*3+dimension] += -dt*cudaD[cudaTypes[particleNumber]]*cudaForces[particleNumber*3+dimension]/KB/T + sqrt(2*cudaD[cudaTypes[particleNumber]]*dt)*hiprand_normal( &localState );
            //cudaCoords[particleNumber*3+dimension] += -dt*cudaD[cudaTypes[particleNumber]]*cudaForces[particleNumber*3+dimension]/KB/T ;
            //cudaCoords[particleNumber*3+dimension] += cudaForces[particleNumber*3+dimension] ;
            cudaForces[particleNumber*3+dimension]=0.0f;
            /// periodic boundary condition
            while(cudaCoords[3*particleNumber+dimension]>cudaBoxSize[dimension*2+1]){cudaCoords[3*particleNumber+dimension]=cudaCoords[3*particleNumber+dimension]-(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2+0]);}
            while(cudaCoords[3*particleNumber+dimension]<cudaBoxSize[dimension*2+0]){cudaCoords[3*particleNumber+dimension]=cudaCoords[3*particleNumber+dimension]+(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2+0]);}
        }
        /// lattice field changed?
        int newVoxel=   ((int)floor((cudaCoords[3*particleNumber+2]-cudaBoxSize[4])/maxCutoff)%cudaLatticeSize[2])*cudaLatticeSize[0]*cudaLatticeSize[1]
                    +((int)floor((cudaCoords[3*particleNumber+1]-cudaBoxSize[2])/maxCutoff)%cudaLatticeSize[1])*cudaLatticeSize[0]
                    +((int)floor((cudaCoords[3*particleNumber+0]-cudaBoxSize[0])/maxCutoff)%cudaLatticeSize[0]);

        /// apply voxel-changes ...
        if(newVoxel!=oldVoxel){
            bool leaveLoop = false;
            /// delete form old list
            while(!leaveLoop){
                /// Lock
                if(atomicExch(&(cudaSemaphore[oldVoxel]),1)==0){
                    int prev=cudaNeighborList[2*particleNumber];
                    int next=cudaNeighborList[2*particleNumber+1];
                    cudaNeighborList[2*prev+1]=next;
                    cudaNeighborList[2*next]=prev;
                    /// was this partilce begin of the linked list?
                    if(cudaNeighborListBegins[oldVoxel]==particleNumber){
                        /// was the particle the only one in this field?
                        if(cudaNeighborList[2*particleNumber]==particleNumber){
                            cudaNeighborListBegins[oldVoxel]=-1;
                        }
                        else{
                            cudaNeighborListBegins[oldVoxel]=cudaNeighborList[2*particleNumber+1];
                        }
                    }
                    leaveLoop=true;
                    /// unLock
                    atomicExch(&(cudaSemaphore[oldVoxel]),0);
                }
            }
            leaveLoop = false;
            /// push ontop of the new list
            while(!leaveLoop){
                /// Lock
                if(atomicExch(&(cudaSemaphore[newVoxel]),1)==0){
                    /// is new list empty?
                    if(cudaNeighborListBegins[newVoxel]!=-1){/// no
                        cudaNeighborList[2*particleNumber]=cudaNeighborList[2*cudaNeighborListBegins[newVoxel]];
                        cudaNeighborList[2*particleNumber+1]=cudaNeighborListBegins[newVoxel];
                        cudaNeighborList[2*cudaNeighborList[2*cudaNeighborListBegins[newVoxel]]+1]=particleNumber;;
                        cudaNeighborList[2*cudaNeighborListBegins[newVoxel]]=particleNumber;
                        cudaNeighborListBegins[newVoxel]=particleNumber;
                    }
                    else{/// first one in new list
                        cudaNeighborList[2*particleNumber+1]=particleNumber;
                        cudaNeighborList[2*particleNumber]=particleNumber;
                        cudaNeighborListBegins[newVoxel]=particleNumber;
                    }
                    leaveLoop=true;
                    /// unLock
                    atomicExch(&(cudaSemaphore[newVoxel]),0);
                }
            }
        }

        globalRandStates[particleNumber] = localState;
    }
    return;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed, int n ){
    int id=blockIdx.x * blockDim.x + threadIdx.x;
    if(id<n){
        hiprand_init ( seed, id, 0, &state[id] );
    }
}


/// pos force radii forceconst types todo links linkbegins
__device__ void lennardJones(){

    return;
}
