#include "hip/hip_runtime.h"

# include <ReaDDyGPU.hpp>
# include <stdlib.h>
# include <stdio.h>
# include <time.h>
# include <math.h>
# include <vector>
# include <hip/hip_runtime.h>
# include <hiprand.h>
# include <hip/hip_runtime.h>
# include <hiprand/hiprand_kernel.h>
# include <sm_11_atomic_functions.h>
#include <initializer_list>

__global__ void update(float* cudaCoords, float* cudaForces, int* cudaTypes, float* cudaD, int * cudaNeighborList, int * cudaNeighborListBegins, float * cudaBoxSize, int * cudaSemaphore, hiprandState* globalRandStates, float dt, int numberParticles, float KB, float T, float maxCutoff, int * latticeSize);
__global__ void orderOne(float* cudaCoords, float* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, float * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderOnePotentialsMatrix, CudaOrderOnePotential * cudaCudaOrderOnePotentials, int numberOfOrderOnePotentials, int numberOfParticleTypes, float * cudaParticleRadiiMatrix);
__global__ void orderTwo(float* cudaCoords, float* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, float * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderTwoPotentialsMatrix, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfOrderTwoPotentials, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix);
__global__ void groups(float* cudaCoords, float* cudaForces, int* cudaTypes, int numberParticles, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix, CudaOrderTwoPotential * cudaCudaGroupPotentials, GroupPart * cudaIndividualGroups, int numberOfIndividualGroups, float * cudaBoxSize);
__device__ void getNeighbors(int particleNumber, int * todo, float* cudaCoords, int * cudaLatticeSize, float * cudaBoxSize, int maxCutoff);
__device__ void calculateOrderTwoPotential(int particleNumber, int interactingParticle, int orderTwoPotentialNr, float r, float* cudaCoords, float* cudaForces, int* cudaTypes, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix);
__global__ void calculateRDF(int * cudaRDFMatrix, float* cudaCoords, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, float * cudaBoxSize, int numberParticles, int maxCutoff, int numberOfParticleTypes, int numberOfBins);
__global__ void setup_kernel ( hiprandState * state, unsigned long seed, int n );

__global__ void warpOrderTwo(int warpsize, float* cudaCoords, float* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, float * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderTwoPotentialsMatrix, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfOrderTwoPotentials, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix);
__device__ void warpCalculateOrderTwoPotential(float * particleCoord, float * particleForce, int particleType, float * interactingParticleCoord, float * interactingParticleForce, int interactingParticleType, int orderTwoPotentialNr, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, float * cudaBoxSize, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix);

CudaSimulation::CudaSimulation(Simulation* simulation){

    this->simulation = simulation;

}

/*__device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old); return __longlong_as_double(old);
}*/
__device__ float atomicCasAdd(float* address, float val) {
    unsigned int* address_as_ui = (unsigned int*)address;
    unsigned int old = *address_as_ui, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ui, assumed, __float_as_int(val + __int_as_float(assumed)));
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    return __int_as_float(old);
}
///Read more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#ixzz3GDRkYOfX


CudaOrderOnePotential toCudaOrderOnePotential(OrderOnePotential* orderOnePotential){
    CudaOrderOnePotential cudaOrderOnePotential = CudaOrderOnePotential();
    if(orderOnePotential->type.compare("DISK")==0){
        DiskPotential * diskPotential = reinterpret_cast<DiskPotential*>(orderOnePotential);
        cudaOrderOnePotential.type=diskPotential->typeID;
        cudaOrderOnePotential.subtype=diskPotential->subtypeID;
        cudaOrderOnePotential.forceConst=diskPotential->forceConst;
        std::copy ( diskPotential->center, diskPotential->center+3, cudaOrderOnePotential.origin );
        std::copy ( diskPotential->normal, diskPotential->normal+3, cudaOrderOnePotential.normal );
        cudaOrderOnePotential.radius=diskPotential->radius;
    }
    else if(orderOnePotential->type.compare("CYLINDER")==0){
        CylinderPotential * cylinderPotential = reinterpret_cast<CylinderPotential*>(orderOnePotential);
        cudaOrderOnePotential.type=cylinderPotential->typeID;
        cudaOrderOnePotential.subtype=cylinderPotential->subtypeID;
        cudaOrderOnePotential.forceConst=cylinderPotential->forceConst;
        std::copy ( cylinderPotential->center, cylinderPotential->center+3, cudaOrderOnePotential.origin );
        std::copy ( cylinderPotential->normal, cylinderPotential->normal+3, cudaOrderOnePotential.normal );
        cudaOrderOnePotential.radius=cylinderPotential->radius;
        cudaOrderOnePotential.height=cylinderPotential->height;
    }
    else if(orderOnePotential->type.compare("SPHERE")==0){
        SpherePotential * spherePotential = reinterpret_cast<SpherePotential*>(orderOnePotential);
        cudaOrderOnePotential.type=spherePotential->typeID;
        cudaOrderOnePotential.subtype=spherePotential->subtypeID;
        cudaOrderOnePotential.forceConst=spherePotential->forceConst;
        std::copy ( spherePotential->center, spherePotential->center+3, cudaOrderOnePotential.origin );
        cudaOrderOnePotential.radius=spherePotential->radius;
    }
    else if(orderOnePotential->type.compare("BOX")==0 || orderOnePotential->type.compare("CUBE")==0){
        BoxPotential * boxPotential = reinterpret_cast<BoxPotential*>(orderOnePotential);
        cudaOrderOnePotential.type=boxPotential->typeID;
        cudaOrderOnePotential.subtype=boxPotential->subtypeID;
        cudaOrderOnePotential.forceConst=boxPotential->forceConst;
        std::copy ( boxPotential->origin, boxPotential->origin+3, cudaOrderOnePotential.origin );
        std::copy ( boxPotential->extension, boxPotential->extension+3, cudaOrderOnePotential.extension );
    }
    return cudaOrderOnePotential;
}

CudaOrderTwoPotential toCudaOrderTwoPotential(OrderTwoPotential* orderTwoPotential){
    CudaOrderTwoPotential cudaOrderTwoPotential = CudaOrderTwoPotential();
    if(orderTwoPotential->type.compare("HARMONIC")==0){
        HarmonicPotential * harmonicPotential = reinterpret_cast<HarmonicPotential*>(orderTwoPotential);
        cudaOrderTwoPotential.type=harmonicPotential->typeID;
        cudaOrderTwoPotential.subtype=harmonicPotential->subtypeID;
        cudaOrderTwoPotential.forceConst = harmonicPotential->forceConst;
    }
    else if(orderTwoPotential->type.compare("WEAK_INTERACTION_HARMONIC")==0){
        WeakInteractionHarmonicPotential * weakInteractionHarmonicPotential = reinterpret_cast<WeakInteractionHarmonicPotential*>(orderTwoPotential);
        cudaOrderTwoPotential.type=weakInteractionHarmonicPotential->typeID;
        cudaOrderTwoPotential.subtype=weakInteractionHarmonicPotential->subtypeID;
        cudaOrderTwoPotential.forceConst=weakInteractionHarmonicPotential->forceConst;
        cudaOrderTwoPotential.length=weakInteractionHarmonicPotential->length;
        cudaOrderTwoPotential.depth=weakInteractionHarmonicPotential->depth;
    }
    return cudaOrderTwoPotential;
}

int CudaSimulation::initialize(){

    int numberOfCudaDevices = 0;
    hipGetDeviceCount(&numberOfCudaDevices);

    if(numberOfCudaDevices==0){
        cout << "no cuda device availible" << endl;
        return 1;
    }
    if(simulation->testmode)
        cout << endl << endl << numberOfCudaDevices << " cuda devices found" << endl << endl;

    for(int i=0; i<numberOfCudaDevices; ++i){

        hipSetDevice(i);
        struct hipDeviceProp_t prop;

        hipGetDeviceProperties(&prop, i);
        if(simulation->testmode){
            cout << prop.name << endl;
            cout << "compute capability: " << prop.major << "." << prop.minor << endl;

            cout << "total global Memory: " << (float)prop.totalGlobalMem/1024.0f/1024.0f/1024.0f << "GB" << endl;
            cout << "shared memory per block: " << (float)prop.sharedMemPerBlock/1024.0f << "KB" << endl;
            cout << "Registers per block: " << (float)prop.regsPerBlock << " a 32-bit -> " <<(float)prop.regsPerBlock*32 << " bit" << endl;
            cout << "total constant memory: " << (float)prop.totalConstMem/1024.0f << "KB" << endl;
            cout << "memory clock rate: " << prop.memoryClockRate << "Hz" << endl;
            cout << "memory bus width: " << prop.memoryBusWidth << "bits" << endl;

            cout << "multi processors: " << prop.multiProcessorCount << endl;
            cout << "clock rate: " << prop.clockRate << "Hz" << endl;

            cout << "warpsize: " << prop.warpSize << endl;
            cout << "max threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << endl;
            cout << "max threads dim: " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << endl;
            cout << "max grid size: " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << endl;
            cout << endl;
        }
    }

    /// ////////////////////////////////////////////////////////////////////////
    cudaDevice = simulation->cudaDevice;
    numberOfThreads = 128;
    /// ////////////////////////////////////////////////////////////////////////

    if(simulation->testmode)
        cout << "cuda device " << cudaDevice << endl;
    hipSetDevice(cudaDevice);
    //hipDeviceReset();


    cout << " test "  << endl;
     hipFree(0) ;
     cout << "testend" << endl;






    gridSize = (simulation->numberParticles/numberOfThreads)+1;
    blockSize = numberOfThreads;
    //gridSize = 10;
    //blockSize = 10;

    if(simulation->testmode)
        cout << "use " << gridSize<< " blocks (grid size) and " << blockSize << " threads (block size) each" << endl;

    maxCutoff = simulation->maxCutoff;
    boxSize = new float[6];
    for(int i=0; i<6; ++i){
        boxSize[i] = simulation->latticeBounds[i];
    }

if(simulation->testmode)
    cout << "initialize cuda random variables" << endl;

    /// initialize cuRand
    hipMalloc ( (void**)&globalRandStates, simulation->numberParticles * sizeof( hiprandState ) );
    if(simulation->testmode){
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "random variable allocation, cuda error: %s\n",hipGetErrorString(error ));
            return 1;
        }
    }
    /// setup seeds
    setup_kernel <<< gridSize, blockSize >>> ( globalRandStates, time(NULL), simulation->numberParticles );
    if(simulation->testmode){
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "random variable initialization, cuda error: %s\n",hipGetErrorString(error ));
            return 1;
        }
    }

if(simulation->testmode)
    cout << "initialize several cuda arrays" << endl;

    /// Coords
    hipMalloc((void**)&cudaCoords,( simulation->numberParticles * 3 * sizeof ( float ) ));
    copyPosToDevice();

    /// Forces
    hipMalloc((void**)&cudaForces,( simulation->numberParticles * 3 * sizeof ( float ) ));
    hipMemset( cudaForces,(float)0, ( simulation->numberParticles * 3 * sizeof ( float ) ));

    /// Diffusion const.
    float * hostDiffConst;
    hostDiffConst = new float[simulation->particleTypes.size()];
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        hostDiffConst[i]=simulation->particleTypes[i].D;
    }
    hipMalloc((void**)&cudaD,( simulation->particleTypes.size() * sizeof ( float ) ));
    hipMemcpy(cudaD, hostDiffConst, ( simulation->particleTypes.size() * sizeof ( float ) ), hipMemcpyHostToDevice);

    /// types
    hipMalloc((void**)&cudaTypes,( simulation->numberParticles * sizeof ( int ) ));
    hipMemcpy(cudaTypes, simulation->types, ( simulation->numberParticles * sizeof ( int ) ), hipMemcpyHostToDevice);

    if(createNeighborList()!=0){
        cout <<"neigborlist building problem" << endl;
        return 1;
    }

    hipMemcpy(cudaNeighborList, hostNeighborList, ( simulation->numberParticles * 2 * sizeof ( int ) ), hipMemcpyHostToDevice);
    hipMemcpy(cudaNeighborListBegins, hostNeighborListBegins, ( numberOfLatticeFields * sizeof ( int ) ), hipMemcpyHostToDevice);

    hipMalloc ((void**)&cudaBoxSize, ( 6 * sizeof ( float ) ));
    hipMemcpy( cudaBoxSize, boxSize, ( 6 * sizeof ( float ) ), hipMemcpyHostToDevice);
    hipMalloc (   (void**)&cudaLatticeSize, ( 3 * sizeof ( int ) ));
    hipMemcpy(cudaLatticeSize, latticeSize, ( 3 * sizeof ( int ) ), hipMemcpyHostToDevice);

    /// cudaSemaphores for the lattice fields
    hipMalloc((void**)&cudaSemaphore,( numberOfLatticeFields * sizeof ( int ) ));
    hipMemset( cudaSemaphore,(int)0, ( numberOfLatticeFields * sizeof ( int ) ));

if(simulation->testmode)
    cout << "initialize cuda order one potentials" << endl;

    /// Matrix for order one potentials = matrix[pot][types] = matrix[simulation->orderOnePotentials.size()][simulation->particleTypes.size()]
    int orderOnePotentialsMatrixSize = simulation->particleTypes.size() * simulation->orderOnePotentials.size();
    hostOrderOnePotentialsMatrix = new int[orderOnePotentialsMatrixSize];
    for(int i=0; i<simulation->orderOnePotentials.size(); ++i){
        for(int j=0; j<simulation->particleTypes.size(); ++j){
            hostOrderOnePotentialsMatrix[i*simulation->particleTypes.size()+j]=0;
        }
        for(int j=0; j<simulation->orderOnePotentials[i]->affectedParticleTypeIds.size(); ++j){
            hostOrderOnePotentialsMatrix[i*simulation->particleTypes.size()+simulation->orderOnePotentials[i]->affectedParticleTypeIds[j]]=1;
        }
    }
    hipMalloc((void**)&cudaOrderOnePotentialsMatrix,( orderOnePotentialsMatrixSize * sizeof ( int ) ));
    hipMemcpy(cudaOrderOnePotentialsMatrix, hostOrderOnePotentialsMatrix, ( orderOnePotentialsMatrixSize * sizeof ( int ) ), hipMemcpyHostToDevice);

    /// create cuda order one pot
    hostCudaOrderOnePotentials = new CudaOrderOnePotential[simulation->orderOnePotentials.size()];
    for(int i=0; i<simulation->orderOnePotentials.size(); ++i){
        hostCudaOrderOnePotentials[i] = toCudaOrderOnePotential(simulation->orderOnePotentials[i]);
    }
    hipMalloc((void**)&cudaCudaOrderOnePotentials,( simulation->orderOnePotentials.size() * sizeof ( CudaOrderOnePotential ) ));
    hipMemcpy(cudaCudaOrderOnePotentials, hostCudaOrderOnePotentials, ( simulation->orderOnePotentials.size() * sizeof ( CudaOrderOnePotential ) ), hipMemcpyHostToDevice);

if(simulation->testmode)
    cout << "initialize cuda order two potentials" << endl;

    /// Lookup for order two potentials
    int numberOfParticleTypes = simulation->particleTypes.size();
    int orderTwoPotentialsMatrixSize = numberOfParticleTypes * numberOfParticleTypes * simulation->orderTwoPotentials.size();
    hostOrderTwoPotentialsMatrix = new int[orderTwoPotentialsMatrixSize];
    for(int numberParticleTypes1=0; numberParticleTypes1<numberOfParticleTypes; ++numberParticleTypes1){
        //cout << numberParticleTypes1 << endl;
        for(int numberParticleTypes2=0; numberParticleTypes2<numberOfParticleTypes; ++numberParticleTypes2){
            //cout << " " << numberParticleTypes2 << endl;
            for(int numberOrderTwoPotentials=0; numberOrderTwoPotentials<simulation->orderTwoPotentials.size(); ++numberOrderTwoPotentials){
                //cout << "  " << numberOrderTwoPotentials << endl;
                hostOrderTwoPotentialsMatrix[numberParticleTypes1*numberOfParticleTypes+numberParticleTypes2*numberOfParticleTypes+numberOrderTwoPotentials]=0;
            }
        }
    }
    for(int orderTwoPotential=0; orderTwoPotential<simulation->orderTwoPotentials.size(); ++orderTwoPotential){
        for(int i=0; i<simulation->orderTwoPotentials[orderTwoPotential]->affectedParticleTypeIdPairs.size(); i+=2){
            //cout << " " << i << endl;
            int particleType1= simulation->orderTwoPotentials[orderTwoPotential]->affectedParticleTypeIdPairs[i];
            int particleType2= simulation->orderTwoPotentials[orderTwoPotential]->affectedParticleTypeIdPairs[i+1];
            hostOrderTwoPotentialsMatrix[particleType1*numberOfParticleTypes+particleType2*numberOfParticleTypes+orderTwoPotential]=1;
            hostOrderTwoPotentialsMatrix[particleType2*numberOfParticleTypes+particleType1*numberOfParticleTypes+orderTwoPotential]=1;
        }
    }

    hipMalloc((void**)&cudaOrderTwoPotentialsMatrix,( orderTwoPotentialsMatrixSize * sizeof ( int ) ));
    hipMemcpy(cudaOrderTwoPotentialsMatrix, hostOrderTwoPotentialsMatrix, ( orderTwoPotentialsMatrixSize * sizeof ( int ) ), hipMemcpyHostToDevice);

    /// create cuda order two pot
    hostCudaOrderTwoPotentials = new CudaOrderTwoPotential[simulation->orderTwoPotentials.size()];
    for(int i=0; i<simulation->orderTwoPotentials.size(); ++i){
        hostCudaOrderTwoPotentials[i] = toCudaOrderTwoPotential(simulation->orderTwoPotentials[i]);
    }
    hipMalloc((void**)&cudaCudaOrderTwoPotentials,( simulation->orderTwoPotentials.size() * sizeof ( CudaOrderTwoPotential ) ));
    hipMemcpy(cudaCudaOrderTwoPotentials, hostCudaOrderTwoPotentials, ( simulation->orderTwoPotentials.size() * sizeof ( CudaOrderTwoPotential ) ), hipMemcpyHostToDevice);

if(simulation->testmode)
    cout << "initialize cuda particle radii and collision radii matrix" << endl;

    /// create cuda paritcle radii matix (for order one ptoentials
    hostParticleRadiiMatrix = new float[simulation->particleTypes.size()];
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        hostParticleRadiiMatrix[i]=simulation->particleTypes[i].defaultRadius;
        //cout << hostParticleRadiiMatrix[i] << endl;
    }
    hipMalloc((void**)&cudaParticleRadiiMatrix,( (simulation->particleTypes.size()) * sizeof ( float ) ));
    hipMemcpy(cudaParticleRadiiMatrix, hostParticleRadiiMatrix, ( simulation->particleTypes.size() * sizeof ( float ) ), hipMemcpyHostToDevice);

    /// create cuda collision radii matix for order two potentials
    hostCollisionRadiiMatrix = new float[(simulation->particleTypes.size())*simulation->particleTypes.size()];
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=0; j<simulation->particleTypes.size(); ++j){
            hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j] = simulation->particleTypes[i].radiiMatrix[j];
        }
        for(int j=0; j<simulation->particleTypes.size(); ++j){
            hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j] = hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j]== 0 ? simulation->particleTypes[i].defaultRadius : hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j];
        }
    }
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=i; j<simulation->particleTypes.size(); ++j){
            float x = hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j]+hostCollisionRadiiMatrix[j*simulation->particleTypes.size()+i];
            hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j] = x;
            hostCollisionRadiiMatrix[j*simulation->particleTypes.size()+i] = x;
        }
    }
    /*for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=0; j<simulation->particleTypes.size(); ++j){
            cout << hostCollisionRadiiMatrix[i*simulation->particleTypes.size()+j] << " " ;
        }
        cout << endl;
    }*/
    hipMalloc((void**)&cudaCollisionRadiiMatrix,( (simulation->particleTypes.size())*simulation->particleTypes.size() * sizeof ( float ) ));
    hipMemcpy(cudaCollisionRadiiMatrix, hostCollisionRadiiMatrix, ( (simulation->particleTypes.size())*simulation->particleTypes.size() * sizeof ( float ) ), hipMemcpyHostToDevice);

if(simulation->testmode)
    cout << "initialize cuda group potentials" << endl;

    /// create cuda group potentials
    int numberOfGroupPotentials=0;
    for(int i=0; i<simulation->groups.size(); ++i){
        numberOfGroupPotentials += simulation->groups[i].potentials.size();
    }
    hostCudaGroupPotentials = new CudaOrderTwoPotential[numberOfGroupPotentials];
    int continuousForceNumber=0;
    for(int i=0; i<simulation->groups.size(); ++i){
        for(int j=0; j<simulation->groups[i].potentials.size(); ++j, ++continuousForceNumber){
            hostCudaGroupPotentials[continuousForceNumber] = toCudaOrderTwoPotential(simulation->groups[i].potentials[j]);
//cout <<"group "  << i << " force "<< hostCudaGroupPotentials[continuousForceNumber].type << endl;
        }
    }
    hipMalloc((void**)&cudaCudaGroupPotentials,( numberOfGroupPotentials * sizeof ( CudaOrderTwoPotential ) ));
    hipMemcpy(cudaCudaGroupPotentials, hostCudaGroupPotentials, ( numberOfGroupPotentials * sizeof ( CudaOrderTwoPotential ) ), hipMemcpyHostToDevice);

    /// list of particle pairs with potential calculations from a group
    vector<GroupPart> groupParts = vector<GroupPart>();
    continuousForceNumber=0;
    for(int i=0; i<simulation->groups.size(); ++i){
        for(int j=0; j<simulation->groups[i].individualGroups.size(); ++j){
            //for(int k=0; k<simulation->groups[i].individualGroups[j].size(); ++k){
            /// TODO: later for more than two!!
                GroupPart groupPart;
                groupPart.particle1=simulation->groups[i].individualGroups[j][0];
                groupPart.particle2=simulation->groups[i].individualGroups[j][1];
                groupPart.groupPot=continuousForceNumber;
                groupParts.push_back(groupPart);
//cout << simulation->groups[i].individualGroups[j][0] << " " <<simulation->groups[i].individualGroups[j][1] <<" " << continuousForceNumber<< endl;
            //}
            //++continuousForceNumber;
        }
    }
    hostIndividualGroups = &groupParts[0];
    numberOfIndividualGroups = groupParts.size();
    /*for(int i=0; i<numberOfIndividualGroups; ++i){
        cout << hostIndividualGroups[i].particle1 << " " << hostIndividualGroups[i].particle2 << " " << hostIndividualGroups[i].groupPot << " " << hostCudaGroupPotentials[hostIndividualGroups[i].groupPot].forceConst << " " << hostCudaGroupPotentials[hostIndividualGroups[i].groupPot].type << " " << hostCudaGroupPotentials[hostIndividualGroups[i].groupPot].subtype << endl;
    }*/
    hipMalloc((void**)&cudaIndividualGroups,( groupParts.size() * sizeof ( GroupPart ) ));
    hipMemcpy(cudaIndividualGroups, hostIndividualGroups, ( groupParts.size() * sizeof ( GroupPart ) ), hipMemcpyHostToDevice);

    /// Matrix for RDF calculation
    /// carefull about reactions! !!!!!!!!!!!!!!!!!!!!!!!!!!
    if(simulation->RDFrequired>0){
        hostRDFMatrix = new int[simulation->particleTypes.size()*simulation->particleTypes.size()*simulation->numberOfRDFBins];
        hipMalloc( (void**)&cudaRDFMatrix,  ( simulation->particleTypes.size() * simulation->particleTypes.size() * simulation->numberOfRDFBins *  sizeof ( int ) ));
        hipMemset( cudaRDFMatrix,(int)0, ( simulation->particleTypes.size() * simulation->particleTypes.size() * simulation->numberOfRDFBins * sizeof ( int ) ));
    }

    hipError_t error = hipGetLastError();
    if ( hipSuccess != error ){
        printf( "cuda error during initialization: %s\n",hipGetErrorString(error) );
        return 1;
    }

if(simulation->testmode)
    cout << "initialization done" << endl;

    return 0;
}

int CudaSimulation::copyRDFMatrix(){

    /// copy from GPU
    hipMemcpy(hostRDFMatrix, cudaRDFMatrix, simulation->particleTypes.size() * simulation->particleTypes.size() * simulation->numberOfRDFBins * sizeof ( int ), hipMemcpyDeviceToHost);

    if(simulation->testmode){
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "cuda error: %s\n",hipGetErrorString(error) );
            return 1;
        }
    }

    /// copy to simulation and normalize
    for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=0; j <simulation->particleTypes.size(); ++j){
            for(int k=0; k<simulation->numberOfRDFBins; ++k){
                simulation->RDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] += (float)hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k]/(float)simulation->numberOfParticlesPerType[i]/(float)simulation->numberOfParticlesPerType[j];
            }
        }
    }

    return 0;
}
int CudaSimulation::copyRDFMatrixToSimulation(){
    return 0;
}

int CudaSimulation::normalizeRDFFRame(){
    /*for(int i=0; i<simulation->particleTypes.size(); ++i){
        for(int j=0; j <simulation->particleTypes.size(); ++j){
            for(int k=0; k<simulation->numberOfRDFBins; ++k){
                /// normalize over particle numbers
                cout << i << "x" << j << "(" << k << "): "<< hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] << " / " <<  simulation->numberOfParticlesPerType[j] << " / " <<  simulation->numberOfParticlesPerType[i] << endl;
                cout << "->" << (float)hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k]/(float)simulation->numberOfParticlesPerType[i]/(float)simulation->numberOfParticlesPerType[j] << endl;
                //hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] = hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k]/simulation->numberOfParticlesPerType[i]/simulation->numberOfParticlesPerType[j];
                /// normalize 2D
                //hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] = hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k];
                /// normalize 3D
                //hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k] = hostRDFMatrix[i*simulation->particleTypes.size()*simulation->numberOfRDFBins+j*simulation->numberOfRDFBins+k];
            }
        }
    }*/
    return 0;
}

int CudaSimulation::callRDFCalculation(){

    //cout << "RDF" << endl;
    calculateRDF<<<gridSize,blockSize>>>(cudaRDFMatrix, cudaCoords, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, simulation->numberParticles, maxCutoff, simulation->particleTypes.size(), simulation->numberOfRDFBins);

    if(simulation->testmode){
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "RDF, cuda error: %s\n",hipGetErrorString(error ));
            return 1;
        }
    }
    ++simulation->numberOfRDFFrames;
    return 0;
}

int CudaSimulation::createNeighborList(){

    numberOfLatticeFields = (boxSize[1]-boxSize[0])/maxCutoff*(boxSize[3]-boxSize[2])/maxCutoff*(boxSize[5]-boxSize[4])/maxCutoff;
    latticeSize = new int[3];
    latticeSize[0] = (boxSize[1]-boxSize[0])/maxCutoff;
    latticeSize[1] = (boxSize[3]-boxSize[2])/maxCutoff;
    latticeSize[2] = (boxSize[5]-boxSize[4])/maxCutoff;
    hipMalloc((void**)&cudaNeighborList,( simulation->numberParticles * 2 * sizeof ( int ) ));
    hipMalloc((void**)&cudaNeighborListBegins,( numberOfLatticeFields * sizeof ( int ) ));

    hostNeighborList = new int[simulation->numberParticles * 2];
    hostNeighborListBegins= new int[numberOfLatticeFields];

    for(int i=0; i<numberOfLatticeFields; ++i){
        hostNeighborListBegins[i]=-1;
    }
    if(simulation->testmode){
        cout << "lattice informations:  " << endl;
        cout << "simulation size x[nm]: " << boxSize[1]-boxSize[0] << endl;
        cout << "simulation size y[nm]: " << boxSize[3]-boxSize[2] << endl;
        cout << "simulation size z[nm]: " << boxSize[5]-boxSize[4] << endl;
        cout << "number of voxels:      " << numberOfLatticeFields << endl;
        cout << "voxel edge length:     " << maxCutoff << endl;
        cout << "lattice size x:        " << latticeSize[0] << endl;
        cout << "lattice size y:        " << latticeSize[1] << endl;
        cout << "lattice size z:        " << latticeSize[2] << endl << endl;
    }

    for(int i=0; i<simulation->numberParticles; ++i){

        int field=((int)floor((simulation->coords[3*i+2]-boxSize[4])/maxCutoff)%latticeSize[2])*latticeSize[0]*latticeSize[1]
                 +((int)floor((simulation->coords[3*i+1]-boxSize[2])/maxCutoff)%latticeSize[1])*latticeSize[0]
                 +((int)floor((simulation->coords[3*i+0]-boxSize[0])/maxCutoff)%latticeSize[0]);

        if(field<0 || field>numberOfLatticeFields){
            cout << "particle is out of the Box: " << i << " [" <<simulation->coords[3*i+0] << ", " << simulation->coords[3*i+1] << ", " << simulation->coords[3*i+2] << "]" << endl;
            return 1;
        }

        if(hostNeighborListBegins[field]==-1){
            /// this particle is the first in this field. it is its own predecessor and successor
            hostNeighborListBegins[field]=i;
            hostNeighborList[2*i+1]=i;
            hostNeighborList[2*i]=i;
        }
        else{
            /// x f y -> x p f y
            /// particles successor is the fields first particle
            /// S'(p) = f
            hostNeighborList[2*i+1]=hostNeighborListBegins[field];
            /// sucessor of the first particles predecessor is the particle
            /// S(P(f))=p , P(f)=x -> S'(x)=p
            hostNeighborList[2*hostNeighborList[2*hostNeighborListBegins[field]]+1]=i;
            /// particles predecessor is the predecessor of the fields first particle
            /// P'(p)=P(f)=x
            hostNeighborList[2*i]=hostNeighborList[2*hostNeighborListBegins[field]];
            /// fields first particles new predecessor is the current particle
            /// P'(f)=p
            hostNeighborList[2*hostNeighborListBegins[field]]=i;
            //hostNeighborListBegins[field]=i;
        }
    }

    hipError_t error = hipGetLastError();
    if ( hipSuccess != error ){
        printf( "cuda error: %s\n",hipGetErrorString(error) );
        return 1;
    }

    return 0;
}

int CudaSimulation::testNeighborList(){

    hipMemcpy(hostNeighborList, cudaNeighborList, ( simulation->numberParticles * 2 * sizeof ( int ) ), hipMemcpyDeviceToHost);
    hipMemcpy(hostNeighborListBegins, cudaNeighborListBegins, ( numberOfLatticeFields * sizeof ( int ) ), hipMemcpyDeviceToHost);

    int count = 0;
    int count2 = 0;
    int x;
    for(int i=0; i<numberOfLatticeFields; ++i){
        x=hostNeighborListBegins[i];
        //cout << i << ":" << x << endl;
        if(x!=-1){
            do{
                count++;
                //cout << hostNeighborList[2*x+0] << " " << x << " " << hostNeighborList[2*x+1] << endl;
                x=hostNeighborList[2*x+1];
                if(x==hostNeighborListBegins[i])
                    break;
                //char a;
                //cin >> a;
            }while(true);
        }
        else{
            ++count2;
        }
    }
    cout << "Neighborlist check:" <<  "count: "<< count << "    part num: " << simulation->numberParticles << "     (check 2:" << count2 << " empty fields)"<<  endl;
    if(count!=simulation->numberParticles){
        cout << "Neighborlist broken!" << endl;
        return 1;
    }

    //cout << "Neighborlist okay!" << endl;

    hipError_t error = hipGetLastError();
    if ( hipSuccess != error ){
        printf( "cuda error: %s\n",hipGetErrorString(error) );
        return 1;
    }

    return 0;
}

int CudaSimulation::copyPosToDevice(){

    hipMemcpy(cudaCoords, simulation->coords, simulation->numberParticles * 3 * sizeof(float), hipMemcpyHostToDevice);

    if(simulation->testmode){
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "cuda error: %s\n",hipGetErrorString(error) );
            return 1;
        }
    }
    return 0;
}

int CudaSimulation::copyPosFromDevice(){

    hipMemcpy(simulation->coords, cudaCoords, simulation->numberParticles * 3 * sizeof ( float ), hipMemcpyDeviceToHost);

    if(simulation->testmode){
        hipError_t error = hipGetLastError();
        if ( hipSuccess != error ){
            printf( "cuda error: %s\n",hipGetErrorString(error) );
            return 1;
        }
    }
    return 0;
}


int CudaSimulation::simulate(){


    //cout << "order one" << endl;
    //orderOne<<<1,1>>>(cudaCoords, cudaForces, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, globalRandStates, simulation->numberParticles, maxCutoff, cudaOrderOnePotentialsMatrix, cudaCudaOrderOnePotentials, simulation->orderOnePotentials.size(), simulation->particleTypes.size(), cudaCollisionRadiiMatrix);
    orderOne<<<gridSize,blockSize>>>(cudaCoords, cudaForces, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, globalRandStates, simulation->numberParticles, maxCutoff, cudaOrderOnePotentialsMatrix, cudaCudaOrderOnePotentials, simulation->orderOnePotentials.size(), simulation->particleTypes.size(), cudaParticleRadiiMatrix);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "order one, cuda error: %s\n",hipGetErrorString(error) );
                return 1;
            }
    }
    //cout << "order two" << endl;
    /*orderTwo<<<gridSize,blockSize>>>(cudaCoords, cudaForces, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, globalRandStates, simulation->numberParticles, maxCutoff, cudaOrderTwoPotentialsMatrix, cudaCudaOrderTwoPotentials, simulation->orderTwoPotentials.size(), simulation->particleTypes.size(), cudaCollisionRadiiMatrix);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "order two, cuda error: %s\n",hipGetErrorString(error) );
                return 1;
            }
    }*/
    //cout << "order two" << endl;
    int warpsize=32;
    //cout << (warpsize*3*4*sizeof(float)+(27+warpsize*3+7)*sizeof(int))/1024.0/8.0 << "KB" << endl;
    warpOrderTwo<<<numberOfLatticeFields, warpsize, (warpsize*3*4*sizeof(float)+(27+warpsize*3+7)*sizeof(int))>>>(warpsize, cudaCoords, cudaForces, cudaTypes, cudaNeighborListBegins, cudaNeighborList, cudaLatticeSize, cudaBoxSize, globalRandStates, simulation->numberParticles, maxCutoff, cudaOrderTwoPotentialsMatrix, cudaCudaOrderTwoPotentials, simulation->orderTwoPotentials.size(), simulation->particleTypes.size(), cudaCollisionRadiiMatrix);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "warp order two, cuda error: %s\n",hipGetErrorString(error) );
                return 1;
            }
    }
    //cout << "groups" << endl;
    /// TODO!: use different grid and block size!
    groups<<<gridSize,blockSize>>>(cudaCoords, cudaForces, cudaTypes, simulation->numberParticles, simulation->particleTypes.size(), cudaCollisionRadiiMatrix, cudaCudaGroupPotentials, cudaIndividualGroups, numberOfIndividualGroups, cudaBoxSize);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "groups, cuda error: %s\n",hipGetErrorString(error) );
                return 1;
            }
    }

    //cout << "update" << endl;
    update<<<gridSize,blockSize>>>(  cudaCoords, cudaForces, cudaTypes, cudaD, cudaNeighborList, cudaNeighborListBegins, cudaBoxSize, cudaSemaphore, globalRandStates, simulation->stepSizeInPs, simulation->numberParticles, simulation->boltzmann,  simulation->temperature, maxCutoff, cudaLatticeSize);
    if(simulation->testmode){
        hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if ( hipSuccess != error ){
                printf( "update, cuda error: %s\n",hipGetErrorString(error ));
                return 1;
            }
    }

    return 0;
}


/// /////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// cuda kernels ////////////////////////////////////////////////////////////////////////////////////////////////
/// /////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void orderOne(float* cudaCoords, float* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, float * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderOnePotentialsMatrix, CudaOrderOnePotential * cudaCudaOrderOnePotentials, int numberOfOrderOnePotentials, int numberOfParticleTypes, float * cudaParticleRadiiMatrix){


    int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;

    ////////////////////////////////////////////////////
    bool considerParticleRadius = true;
    //////////////////////////////////


    if(particleNumber<numberParticles){
        hiprandState localState = globalRandStates[particleNumber];

        /// do calculation of forces and maybe reactions here ...

        /// go through all order one potetntials
        for(int orderOnePotential=0; orderOnePotential<numberOfOrderOnePotentials; ++orderOnePotential){
            ///  lookup in matrix whether they apply to the current particle type
            if(cudaOrderOnePotentialsMatrix[orderOnePotential*numberOfParticleTypes+cudaTypes[particleNumber]]==1){
                /// check what kind of potential it is
                if(cudaCudaOrderOnePotentials[orderOnePotential].type==1){/// Disk

                    /// calculation depends on the normal vector. assign x,y and z coordinates to variables
                    int normal, side1, side2;
                    /// normal vector on x axis -> assign x to normal and y and z to the lateral (on Disk) directions
                    if(cudaCudaOrderOnePotentials[orderOnePotential].normal[0]==1){
                        normal=0;side1=1;side2=2;
                    }
                    /// y
                    else if(cudaCudaOrderOnePotentials[orderOnePotential].normal[1]==1){
                        normal=1;side1=0;side2=2;
                    }
                    /// x
                    else {
                        normal=2;side1=1;side2=0;
                    }

                    /// different subtypes
                    if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==1){/// attractive

                        float r = cudaCudaOrderOnePotentials[orderOnePotential].origin[normal]-cudaCoords[3*particleNumber+normal];
                        cudaForces[3*particleNumber+normal]+=-cudaCudaOrderOnePotentials[orderOnePotential].forceConst*r;

                        /// particle radius!
                        r = sqrt(
                                    pow(cudaCoords[3*particleNumber+side1]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side1],2)
                                    +
                                    pow(cudaCoords[3*particleNumber+side2]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side2],2)
                                );
                        if (r > cudaCudaOrderOnePotentials[orderOnePotential].radius) {
                            cudaForces[3*particleNumber+side1]+=
                                    -cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                    *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                    /r
                                    *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side1]-cudaCoords[3*particleNumber+side1]);
                            cudaForces[3*particleNumber+side2]+=
                                    -cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                    *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                    /r
                                    *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side2]-cudaCoords[3*particleNumber+side2]);

                        }
                    }
                    else if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==2){/// repulsive
                        // makes no sense ...
                        /*
                        // force along normal vector
                        r = distToDiskPlane;// actual
                        r0 = pRadius;// desired
                        float r_1 = distToCenterWithinDiskPlane - pRadius;
                        float r0_1 = diskRadius;
                        if (r < r0 && r_1 < r0_1) {

                            precompute = (k * (-r0 + r) / r);

                                gradient[0] = gradient[0]+ precompute * ( pointOnDiskPlane[0]-coords1[0]);
                                gradient[1] = gradient[1]+ precompute * ( pointOnDiskPlane[1]-coords1[1]);
                                gradient[2] = gradient[2]+ precompute * ( pointOnDiskPlane[2]-coords1[2]);

                        }*/
                    }
                }/// end Disk
                else if(cudaCudaOrderOnePotentials[orderOnePotential].type==2){/// Cylinder

                        int normal, side1, side2;
                        if(cudaCudaOrderOnePotentials[orderOnePotential].normal[0]==1){normal=0;side1=1;side2=2;}
                        else if(cudaCudaOrderOnePotentials[orderOnePotential].normal[1]==1){normal=1;side1=0;side2=2;}
                        else {normal=2;side1=1;side2=0;}

                        if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==1){/// attractive

                            float r = cudaCudaOrderOnePotentials[orderOnePotential].origin[normal]-cudaCoords[3*particleNumber+normal];
                            if(fabsf(r)>cudaCudaOrderOnePotentials[orderOnePotential].height*0.5)
                            cudaForces[3*particleNumber+normal]+=-cudaCudaOrderOnePotentials[orderOnePotential].forceConst*(fabsf(r)-cudaCudaOrderOnePotentials[orderOnePotential].height*0.5)/fabsf(r)*r;

                            /// particle radius!
                            r = sqrt(
                                        pow(cudaCoords[3*particleNumber+side1]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side1],2)
                                        +
                                        pow(cudaCoords[3*particleNumber+side2]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side2],2)
                                    );
                            if (r > cudaCudaOrderOnePotentials[orderOnePotential].radius) {
                                cudaForces[3*particleNumber+side1]+=
                                        -cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                        *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                        /r
                                        *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side1]-cudaCoords[3*particleNumber+side1]);
                                cudaForces[3*particleNumber+side2]+=
                                        -cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                        *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                        /r
                                        *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side2]-cudaCoords[3*particleNumber+side2]);

                            }
                        }
                        else if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==2){/// repulsive

                            float r = cudaCudaOrderOnePotentials[orderOnePotential].origin[normal]-cudaCoords[3*particleNumber+normal];
                            if(fabsf(r)<cudaCudaOrderOnePotentials[orderOnePotential].height*0.5)
                            cudaForces[3*particleNumber+normal]+=cudaCudaOrderOnePotentials[orderOnePotential].forceConst*(fabsf(r)-cudaCudaOrderOnePotentials[orderOnePotential].height*0.5)/fabsf(r)*r;

                            /// particle radius!
                            r = sqrt(
                                        pow(cudaCoords[3*particleNumber+side1]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side1],2)
                                        +
                                        pow(cudaCoords[3*particleNumber+side2]- cudaCudaOrderOnePotentials[orderOnePotential].origin[side2],2)
                                    );
                            if (r < cudaCudaOrderOnePotentials[orderOnePotential].radius) {
                                cudaForces[3*particleNumber+side1]+=
                                        cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                        *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                        /r
                                        *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side1]-cudaCoords[3*particleNumber+side1]);
                                cudaForces[3*particleNumber+side2]+=
                                        cudaCudaOrderOnePotentials[orderOnePotential].forceConst
                                        *(r-cudaCudaOrderOnePotentials[orderOnePotential].radius)
                                        /r
                                        *(cudaCudaOrderOnePotentials[orderOnePotential].origin[side2]-cudaCoords[3*particleNumber+side2]);

                            }
                        }
                }/// end Cylinder
                else if(cudaCudaOrderOnePotentials[orderOnePotential].type==3){/// Sphere
                        float dist = 0;
                        for(int dim=0; dim<3; ++dim){
                            dist += (cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]-cudaCoords[3*particleNumber+dim])*(cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]-cudaCoords[3*particleNumber+dim]);
                        }
                        dist = sqrt(dist);

                        if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==1 && dist>cudaCudaOrderOnePotentials[orderOnePotential].radius){/// attractive
                            float precompute = cudaCudaOrderOnePotentials[orderOnePotential].forceConst * (dist - cudaCudaOrderOnePotentials[orderOnePotential].radius) / dist;
                            for(int dim=0; dim<3; ++dim){
                                cudaForces[3*particleNumber+dim] += -precompute * (cudaCudaOrderOnePotentials[orderOnePotential].origin[dim] - cudaCoords[3*particleNumber+dim]);
                            }
                        }
                        else if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==2 && dist<cudaCudaOrderOnePotentials[orderOnePotential].radius){/// repulsive
                            float precompute = cudaCudaOrderOnePotentials[orderOnePotential].forceConst * (dist - cudaCudaOrderOnePotentials[orderOnePotential].radius) / dist;
                            for(int dim=0; dim<3; ++dim){
                                cudaForces[3*particleNumber+dim] += -precompute * (cudaCudaOrderOnePotentials[orderOnePotential].origin[dim] - cudaCoords[3*particleNumber+dim]);
                            }
                        }
                }/// end Sphere
                else if(cudaCudaOrderOnePotentials[orderOnePotential].type==4){/// Box
                        if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==1){/// attractive
                            for(int dim=0; dim<3; ++dim){
                                float distToBoxBegin = cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]+cudaCudaOrderOnePotentials[orderOnePotential].extension[dim]-(considerParticleRadius ? cudaParticleRadiiMatrix[cudaTypes[particleNumber]] : 0);
                                if(cudaCoords[3*particleNumber+dim] > distToBoxBegin){
                                    cudaForces[3*particleNumber+dim]+= -cudaCudaOrderOnePotentials[orderOnePotential].forceConst *
                                            (distToBoxBegin - cudaCoords[3*particleNumber+dim]);
                                }
                                else{
                                    float distToBoxEnd = cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]+(considerParticleRadius ? cudaParticleRadiiMatrix[cudaTypes[particleNumber]] : 0);
                                    if(cudaCoords[3*particleNumber+dim] < distToBoxEnd){
                                    cudaForces[3*particleNumber+dim]+= cudaCudaOrderOnePotentials[orderOnePotential].forceConst *
                                            (cudaCoords[3*particleNumber+dim]-distToBoxEnd);
                                    }
                                }
                            }
                        }
                        else if(cudaCudaOrderOnePotentials[orderOnePotential].subtype==2){/// repulsive
                            for(int dim=0; dim<3; ++dim){
                                float distToBoxBegin = cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]+cudaCudaOrderOnePotentials[orderOnePotential].extension[dim]+(considerParticleRadius ? cudaParticleRadiiMatrix[cudaTypes[particleNumber]] : 0);
                                if(cudaCoords[3*particleNumber+dim] > distToBoxBegin){
                                    cudaForces[3*particleNumber+dim] += cudaCudaOrderOnePotentials[orderOnePotential].forceConst *
                                            (distToBoxBegin - cudaCoords[3*particleNumber+dim]);
                                }
                                else{
                                    float distToBoxEnd = cudaCudaOrderOnePotentials[orderOnePotential].origin[dim]-(considerParticleRadius ? cudaParticleRadiiMatrix[cudaTypes[particleNumber]] : 0);
                                    if(cudaCoords[3*particleNumber+dim] < distToBoxEnd){
                                    cudaForces[3*particleNumber+dim] += -cudaCudaOrderOnePotentials[orderOnePotential].forceConst *
                                            (cudaCoords[3*particleNumber+dim]-distToBoxEnd);
                                    }
                                }
                            }
                        }
                }/// end Box
            }/// endif order one potentials matrix
        }/// end iterate over order one potentials

        globalRandStates[particleNumber] = localState;
    }
    return;
}

__device__ void calculateOrderTwoPotential(int particleNumber, int interactingParticle, int orderTwoPotentialNr, float r, float* cudaCoords, float* cudaForces, int* cudaTypes, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix){

    float r0 = cudaCollisionRadiiMatrix[cudaTypes[particleNumber]*numberOfParticleTypes+cudaTypes[interactingParticle]];
    if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].type==1){        /// Harmonic Potential
        float precompute = cudaCudaOrderTwoPotentials[orderTwoPotentialNr].forceConst * (r-r0)/r;
        for(int coord=0; coord<3; ++coord){
            float force = precompute * (cudaCoords[interactingParticle*3+coord]-cudaCoords[particleNumber*3+coord]);
            if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==1 && r>r0){ /// attractive
                atomicCasAdd(&cudaForces[particleNumber*3+coord], -force );
                atomicCasAdd(&cudaForces[interactingParticle*3+coord], force );
            }
            else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==2 && r<r0){ /// repulsive
                atomicCasAdd(&cudaForces[particleNumber*3+coord], -force );
                atomicCasAdd(&cudaForces[interactingParticle*3+coord], force );
            }
            else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==3){ /// spring
                atomicCasAdd(&cudaForces[particleNumber*3+coord], -force );
                atomicCasAdd(&cudaForces[interactingParticle*3+coord], force );
            }
        }
    }
    else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].type==2){     /// Harmonic weak interaction Potential
        float iradius = cudaCudaOrderTwoPotentials[orderTwoPotentialNr].length + r0;
        if (r < iradius && r > r0) {
            float precompute =  ( cudaCudaOrderTwoPotentials[orderTwoPotentialNr].forceConst * (-r0 + r) * (iradius -r));
            for(int coord=0; coord<3; ++coord){
                float force = precompute * (cudaCoords[interactingParticle*3+coord]-cudaCoords[particleNumber*3+coord]);
                atomicCasAdd(&cudaForces[particleNumber*3+coord], -force );
                atomicCasAdd(&cudaForces[interactingParticle*3+coord], force );
            }
        }
    }
}

__device__ void getNeighbors(int particleNumber, int * todo, float* cudaCoords, int * cudaLatticeSize, float * cudaBoxSize, int maxCutoff){

    int x,y,z;
    int field=((int)floor((cudaCoords[3*particleNumber+2]-cudaBoxSize[4])/maxCutoff)%cudaLatticeSize[2])*cudaLatticeSize[0]*cudaLatticeSize[1]
             +((int)floor((cudaCoords[3*particleNumber+1]-cudaBoxSize[2])/maxCutoff)%cudaLatticeSize[1])*cudaLatticeSize[0]
             +((int)floor((cudaCoords[3*particleNumber+0]-cudaBoxSize[0])/maxCutoff)%cudaLatticeSize[0]);

    /// surrounding, for calculation imprtant fields
    /// TODO: CHECK! !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

        for(x=-1; x<2;x++){
            for(y=-1; y<2;y++){
                for(z=-1; z<2;z++){
                    todo[(x+1)+(y+1)*3+(z+1)*9]=
                    (
                        (
                            (field%(cudaLatticeSize[0]))
                            +x+cudaLatticeSize[0]
                        )
                        %cudaLatticeSize[0]
                    )

                    +cudaLatticeSize[0]*
                    (
                        (
                            (int)floorf
                            (
                                (float)(field%(cudaLatticeSize[0]*cudaLatticeSize[1]))
                                /
                                (float)(cudaLatticeSize[0])
                            )
                            +y+cudaLatticeSize[1]
                        )
                        %cudaLatticeSize[1]
                    )

                    +cudaLatticeSize[0]*cudaLatticeSize[1]*
                    (
                        (
                            (int)floorf
                            (
                                (float)(field)
                                /
                                (float)(cudaLatticeSize[0]*cudaLatticeSize[1])
                            )
                            +z+cudaLatticeSize[2]
                        )
                        %cudaLatticeSize[2]
                    );
                }
            }
        }


}

__global__ void orderTwo(float* cudaCoords, float* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, float * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderTwoPotentialsMatrix, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfOrderTwoPotentials, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix){

    int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;

    if(particleNumber<numberParticles){
        //hiprandState localState = globalRandStates[particleNumber];

        int todo[27];

        getNeighbors(particleNumber, todo, cudaCoords, cudaLatticeSize, cudaBoxSize, maxCutoff);

        /// do calculation of forces and maybe reactions here ...

            /** TODO!!:
             *bring matrices to shared mem
             *interaction matrix could contain the specific cutoffs
             *think about parameter storage
             *then dynamik arrays
             *later think about accellerations due to ideas below
             *
             * need:   - interaction matices (radii, forces) for every force (aligned in one array, +array size)
             *      - more parameter? how to store?
             *  data alignment: x,y,z,type,rand?,force?
             *
             * call voxel per warp(n threads)
             * load first n coords in shared mem
             * calculate distances to particles in surrounding fields (always load one particle and calc. n distances)
             *
             * check for periodic boundaries ...
             */


            /// loop over all "todo" fields around the current field
            for(int x=0; x<27; x++){
                /// begin link to the first element from the list of the field
                int interactingParticle=cudaNeighborListBegins[todo[x]];
                if(interactingParticle!=-1){
                    do
                    {
                        //if(interactingParticle!=particleNumber){
                        /// calculating interaction just once, and apply it for both particles
                        if(interactingParticle<particleNumber){

                            float r=0.0f;
                            float rij[3];
                            for (int dim=0;dim<3;dim++){
                                rij[dim]=cudaCoords[3*particleNumber+dim]-cudaCoords[3*interactingParticle+dim];
                                if(rij[dim]>( (cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
                                if(rij[dim]<(-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]+(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
                                r=r+(rij[dim]*rij[dim]);
                            }
                            r=sqrtf(r);

                            for(int orderTwoPotentialNr=0; orderTwoPotentialNr<numberOfOrderTwoPotentials; ++orderTwoPotentialNr){
                                if(cudaOrderTwoPotentialsMatrix[cudaTypes[particleNumber]*numberOfParticleTypes+cudaTypes[interactingParticle]*numberOfParticleTypes+orderTwoPotentialNr]!=0 && r<=maxCutoff){
                                    calculateOrderTwoPotential(particleNumber, interactingParticle, orderTwoPotentialNr, r, cudaCoords, cudaForces, cudaTypes, cudaCudaOrderTwoPotentials, numberOfParticleTypes, cudaCollisionRadiiMatrix);
                                }
                            }
                        }
                        interactingParticle=cudaNeighborList[2*interactingParticle+1];
                    }
                    while(interactingParticle!=cudaNeighborListBegins[todo[x]]);/// loop/list end
                }
            }
        //globalRandStates[particleNumber] = localState;
    }
    return;
}

__device__ void warpGetNeighbors(int field, int * todo, int * cudaLatticeSize){

    if(threadIdx.x<27){
        int x,y,z;

        /// /////////////////////////////////////////////////////////////
        /// modulo is slow!!
        /// /////////////////////////////////////////////////////////////

        x=threadIdx.x/9%3 -1;
        y=threadIdx.x/3%3 -1;
        z=threadIdx.x%3 -1;

        //for(x=-1; x<2;x++){
            //for(y=-1; y<2;y++){
                //for(z=-1; z<2;z++){
                    todo[(x+1)+(y+1)*3+(z+1)*9]=
                    (
                        (
                            (field%(cudaLatticeSize[0]))
                            +x+cudaLatticeSize[0]
                        )
                        %cudaLatticeSize[0]
                    )

                    +cudaLatticeSize[0]*
                    (
                        (
                            (int)floorf
                            (
                                (float)(field%(cudaLatticeSize[0]*cudaLatticeSize[1]))
                                /
                                (float)(cudaLatticeSize[0])
                            )
                            +y+cudaLatticeSize[1]
                        )
                        %cudaLatticeSize[1]
                    )

                    +cudaLatticeSize[0]*cudaLatticeSize[1]*
                    (
                        (
                            (int)floorf
                            (
                                (float)(field)
                                /
                                (float)(cudaLatticeSize[0]*cudaLatticeSize[1])
                            )
                            +z+cudaLatticeSize[2]
                        )
                        %cudaLatticeSize[2]
                    );
                //}
            //}
        //}
    }
}

__device__ void warpCalculateOrderTwoPotential(float * particleCoord, float * particleForce, int particleType, float * interactingParticleCoord, float * interactingParticleForce, int interactingParticleType, int orderTwoPotentialNr, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, float * cudaBoxSize, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix){

    float r=0.0f;
    float rij[3];
    for (int dim=0;dim<3;dim++){
        rij[dim]=particleCoord[dim]-interactingParticleCoord[dim];
        if(rij[dim]>( (cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
        if(rij[dim]<(-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]+(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
        r=r+(rij[dim]*rij[dim]);
    }
    r=sqrtf(r);

    float r0 = cudaCollisionRadiiMatrix[particleType*numberOfParticleTypes+interactingParticleType];
    if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].type==1){        /// Harmonic Potential
        float precompute = cudaCudaOrderTwoPotentials[orderTwoPotentialNr].forceConst * (r-r0)/r;
        for(int dim=0; dim<3; ++dim){
            float force = precompute * (interactingParticleCoord[dim]-particleCoord[dim]);
            if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==1 && r>r0){ /// attractive
                atomicCasAdd(&particleForce[dim], -force );
                atomicCasAdd(&interactingParticleForce[dim], force );
            }
            else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==2 && r<r0){ /// repulsive
                atomicCasAdd(&particleForce[dim], -force );
                atomicCasAdd(&interactingParticleForce[dim], force );
            }
            else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].subtype==3){ /// spring
                atomicCasAdd(&particleForce[dim], -force );
                atomicCasAdd(&interactingParticleForce[dim], force );
            }
        }
    }
    else if(cudaCudaOrderTwoPotentials[orderTwoPotentialNr].type==2){     /// Harmonic weak interaction Potential
        float iradius = cudaCudaOrderTwoPotentials[orderTwoPotentialNr].length + r0;
        if (r < iradius && r > r0) {
            float precompute =  ( cudaCudaOrderTwoPotentials[orderTwoPotentialNr].forceConst * (-r0 + r) * (iradius -r));
            for(int dim=0; dim<3; ++dim){
                float force = precompute * (interactingParticleCoord[dim]-particleCoord[dim]);
                atomicCasAdd(&particleForce[dim], -force );
                atomicCasAdd(&interactingParticleForce[dim], force );
            }
        }
    }
}

__global__ void warpOrderTwo(int warpsize, float* cudaCoords, float* cudaForces, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, float * cudaBoxSize, hiprandState* globalRandStates, int numberParticles, int maxCutoff, int * cudaOrderTwoPotentialsMatrix, CudaOrderTwoPotential * cudaCudaOrderTwoPotentials, int numberOfOrderTwoPotentials, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix){

    //int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;
    //hiprandState localState;
    //hiprandState localState = globalRandStates[particleNumber];

    int thread = threadIdx.x;
    int field = blockIdx.x;
    int particleNumber = cudaNeighborListBegins[field];
    if(particleNumber==-1){
        return;
    }

    /// link to shared memory
    extern __shared__ float s[];

    //__shared__ float localParticleCoords[warpsize*3];
    float * localParticleCoords = (float*)s;
    //__shared__ float localParticleForces[warpsize*3];
    float * localParticleForces = (float*)&localParticleCoords[warpsize*3];
    //__shared__ float localInteractingParticleCoords[warpsize*3];
    float * localInteractingParticleCoords = (float*)&localParticleForces[warpsize*3];
    //__shared__ float localInteractingParticleForces[warpsize*3];
    float * localInteractingParticleForces = (float*)&localInteractingParticleCoords[warpsize*3];

    //__shared__ int todo[27];
    int * todo = (int*)&localInteractingParticleForces[warpsize*3];
    //__shared__ int localParticleTypes[warpsize];
    int * localParticleTypes = (int*)&todo[27];
    //__shared__ int localInteractingParticleTypes[warpsize];
    int * localInteractingParticleTypes = (int*)&localParticleTypes[warpsize];
    //__shared__ int localInteractingParticleNumber[warpsize];
    int * localInteractingParticleNumber = (int*)&localInteractingParticleTypes[warpsize];
    //__shared__ int lastParticle;
    int * lastParticle = (int*)&localInteractingParticleNumber[warpsize];
    //__shared__ int end;
    int * end = (int*)&lastParticle[1];
    //__shared__ int todoEnd;
    int * todoEnd = (int*)&end[1];
    //__shared__ int lastTodoField;
    int * lastTodoField = (int*)&todoEnd[1];
    //__shared__ int lastInteractingParticle;
    int * lastInteractingParticle = (int*)&lastTodoField[1];
    int * numberOfLoadedParticles = (int*)& lastInteractingParticle[1];
    int * numberOfLoadedInteractingParticles = (int*)& numberOfLoadedParticles[1];
    /// sum = (warpsize*3*4*sizeof(float)+(27+warpsize*3+7)*sizeof(int))

    /* may load to shared mem:
     *cudaNeighborListBegins[todo fields]
     *
     *shrink todo to 14
     */

    warpGetNeighbors(field, todo, cudaLatticeSize);
//    if(thread==0){
//        getNeighbors(particleNumber, todo, cudaCoords, cudaLatticeSize, cudaBoxSize, maxCutoff);
//    }

    if(thread==0){
        atomicExch(end, 0);
    }
    /// iterate over particles in this field
    do{
        if(thread==0){
            atomicExch(numberOfLoadedParticles, warpsize);
        }
        /// assign each thread a particle //////////////////////////////////////////////////////////////////
        bool loadedAParticle = true;
        for(int i=1; i<thread; i++){
            if(cudaNeighborList[particleNumber*2+1] == cudaNeighborListBegins[field]){
                if(thread==i){
                    atomicExch(end, 1);
                    atomicExch(numberOfLoadedParticles, i);
                }
                loadedAParticle = false;
                break;
                //return;
            }
            else{
                particleNumber = cudaNeighborList[particleNumber*2+1];
            }
        }
        /// store particle properties local
        if(loadedAParticle){
            for(int dim=0; dim<3; ++dim){
                localParticleCoords[3*thread+dim]=cudaCoords[3*particleNumber+dim];
                localParticleForces[3*thread+dim]=0;
            }
            localParticleTypes[thread]=cudaTypes[particleNumber];
        }

        int x=0;
        int interactingParticleNumber=cudaNeighborListBegins[todo[x]];

        /// for all interacting fields ///////////////////////////////////////////////////////////////////
        if(thread==0){
            atomicExch(todoEnd, 0);
        }
        /// auxillary variable for starting the interacting particle-search
        bool load_first_particle = true;
        do{

//atomicExch(todoEnd, 1);

            if(thread==0){
                atomicExch(numberOfLoadedInteractingParticles, warpsize);
            }
            bool loadedAInteractingParticle = true;
            /// get one interacting particle for all threads
            for(int i=0; i<=thread; i++){
/*                /// first(0) thread gets first particle in list
                /// if it occures again, we had all from this field
                if(interactingParticleNumber == cudaNeighborListBegins[todo[x]] && i!=0){
                    ++x;    /// next todo field
                    if(x<=14){
                        interactingParticleNumber = cudaNeighborListBegins[todo[x]];   /// start at next todo field
                        --i;
                        continue;
                    }
                }
                /// field is empty
                else if(interactingParticleNumber==-1){
                    ++x;    /// next todo field
                    if(x<=14){
                        interactingParticleNumber = cudaNeighborListBegins[todo[x]];   /// start at next todo field
                        --i;
                        continue;
                    }
                }
                else if(x>14){
                    if(thread==i){
                        atomicExch(todoEnd, 1);
                        atomicExch(numberOfLoadedInteractingParticles, i);
                    }
                    loadedAInteractingParticle = false;
                    break;
                }
                //interactingParticleNumber = cudaNeighborList[interactingParticleNumber*2+1];
*/
                /// break condition, if we checked all todo fields
                if(x>13){
                    if(thread==i){
                        atomicExch(todoEnd, 1);
                        atomicExch(numberOfLoadedInteractingParticles, i);
                    }
                    loadedAInteractingParticle = false;
                    break;
                }
                /// empty field
                if(interactingParticleNumber == -1){
                    /// take next field
                    ++x;
                    if(x<=13){
                        interactingParticleNumber = cudaNeighborListBegins[todo[x]];
                        /// if next field is also empty, proceed with next field (in next iteration)
                        if(interactingParticleNumber == -1){
                            --i;    /// this thread loaded no particle, try again
                            continue;
                        }
                        /// if field is not empty, proceed with its first element
                        else{
                            continue;
                        }
                    }
                }
                /// if we reach the begin of the field (first particle in fields list) again (not valid for very first particle)
                else if(interactingParticleNumber == cudaNeighborListBegins[todo[x]] && !load_first_particle){
                    /// take next field
                    ++x;
                    if(x<=13){
                        interactingParticleNumber = cudaNeighborListBegins[todo[x]];
                        /// if next field is also empty, proceed with next field (in next iteration)
                        if(interactingParticleNumber == -1){
                            --i;    /// this thread loaded no particle, try again
                            continue;
                        }
                        /// if field is not empty, proceed with its first element
                        else{
                            continue;
                        }
                    }
                }
                else{
                    interactingParticleNumber = cudaNeighborList[interactingParticleNumber*2+1];
                }
            }
            load_first_particle = false;

            /// store interacting particle properties local
            if(loadedAInteractingParticle){
                for(int dim=0; dim<3; ++dim){
                    localInteractingParticleCoords[3*thread+dim]=cudaCoords[3*interactingParticleNumber+dim];
                    localInteractingParticleForces[3*thread+dim]=0;
                }
                localInteractingParticleTypes[thread]=cudaTypes[interactingParticleNumber];
                localInteractingParticleNumber[thread]=interactingParticleNumber;
            }

            /// calc
            int calculationCycle = 0;
            /// particle != interacting particle!!!
            while(calculationCycle*warpsize+thread<numberOfLoadedParticles[0]*numberOfLoadedInteractingParticles[0]){
            // for(int calculationCycle = 0; calculationCycle<(numberOfLoadedParticles*numberOfLoadedInteractingParticles)/warpsize; ++calculationCycle){
                int particleToCalculate = calculationCycle*warpsize%numberOfLoadedParticles[0];
                int interactingParticleToCalculate = calculationCycle*warpsize%numberOfLoadedInteractingParticles[0];
                for(int orderTwoPotentialNr=0; orderTwoPotentialNr<numberOfOrderTwoPotentials; ++orderTwoPotentialNr){
                    if(cudaOrderTwoPotentialsMatrix[cudaTypes[particleNumber]*numberOfParticleTypes+cudaTypes[interactingParticleNumber]*numberOfParticleTypes+orderTwoPotentialNr]!=0){
                        warpCalculateOrderTwoPotential(&localParticleCoords[particleToCalculate], &localParticleForces[particleToCalculate], localParticleTypes[particleToCalculate], &localInteractingParticleCoords[interactingParticleToCalculate], &localInteractingParticleForces[interactingParticleToCalculate], localInteractingParticleTypes[interactingParticleToCalculate], orderTwoPotentialNr, cudaCudaOrderTwoPotentials, cudaBoxSize, numberOfParticleTypes, cudaCollisionRadiiMatrix);
                    }
                }
                ++calculationCycle;
            }
            if(loadedAInteractingParticle){
                interactingParticleNumber = localInteractingParticleNumber[thread];
                /// store local changes in global mem
                for(int dim=0; dim<3; ++dim){
                    atomicCasAdd(&cudaForces[interactingParticleNumber*3+dim], localInteractingParticleForces[thread+dim]);
                }
            }

            /// just the last thread, with the last interacting particle stores its number and field, so that the threads can start again from there.
            if(thread==warpsize-1){
                /* /// there is a next particle in this field
                if(cudaNeighborList[interactingParticleNumber*2+1]!=cudaNeighborListBegins[x]){
                    /// store the particle and field number
                    atomicExch(lastInteractingParticle, cudaNeighborList[interactingParticleNumber*2+1]);
                    atomicExch(lastTodoField, x);
                    atomicExch(todoEnd, 0);    /// we are not done jet
                }
                /// if we are done with this todo field
                else{
                    atomicExch(todoEnd, 1);    /// we are done, if there are no more particle -> check
                    /// as long, as we have fields to consider (todo)
                    //while(x>27){
                    while(x>14){ // just do first half due to force-symmetry
                        ++x;    /// take next todo field
                        if(cudaNeighborListBegins[todo[x]]==-1)   /// empty?
                            continue;                       /// again. -> take next
                        else{   /// todo field not empty -> store field and first particle
                            atomicExch(lastInteractingParticle, cudaNeighborListBegins[todo[x]]);
                            atomicExch(lastTodoField, x);
                            atomicExch(todoEnd, 0);    /// we are not done jet
                        }
                    }
                }*/
                atomicExch(lastInteractingParticle, interactingParticleNumber);
                atomicExch(lastTodoField, x);
            }
            /// block wise sync
            __syncthreads();
            x=lastTodoField[0];
            interactingParticleNumber=lastInteractingParticle[0];
        }while(todoEnd[0]==0); /// end interacting fields

        if(loadedAParticle){
            /// store locally saved changes global (interacting particles)
            for(int dim=0; dim<3; ++dim){
                //atomicCasAdd(&cudaForces[particleNumber*3+dim], localParticleForces[thread+dim]);
//                atomicCasAdd(&cudaForces[particleNumber*3+dim], localParticleForces[thread+dim]/2); // devide by two, because the forces are computed twice (in this field), due to force symmetry(for the other fields)
            }
        }

        /// if more particle in this field continue
        if(thread==warpsize-1){
            if(cudaNeighborList[particleNumber*2+1]!=cudaNeighborListBegins[field]){
                atomicExch(lastParticle, cudaNeighborList[particleNumber*2+1]);
                atomicExch(end, 0);
            }
            else{
                atomicExch(end, 1);
            }
        }
        /// block wise sync
        __syncthreads();
        particleNumber=lastParticle[0];
    }while(end[0]==0); /// end loop over particles from this field

    /// store locally saved changes global (particles)

    //globalRandStates[particleNumber] = localState;
    return;
}

__global__ void calculateRDF(int * cudaRDFMatrix, float* cudaCoords, int* cudaTypes, int * cudaNeighborListBegins, int * cudaNeighborList, int * cudaLatticeSize, float * cudaBoxSize, int numberParticles, int maxCutoff, int numberOfParticleTypes, int numberOfBins){

    int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;

    if(particleNumber<numberParticles){

        int todo[27];

        getNeighbors(particleNumber, todo, cudaCoords, cudaLatticeSize, cudaBoxSize, maxCutoff);

        /// loop over all "todo" fields around the current field
        for(int x=0; x<27; x++){
            /// begin link to the first element from the list of the field
            int interactingParticle=cudaNeighborListBegins[todo[x]];
            if(interactingParticle!=-1){
                do
                {
                    //if(interactingParticle!=particleNumber){
                    /// calculating interaction just once, and apply it for both particles
                    if(interactingParticle<particleNumber){

                        float r=0.0f;
                        float rij[3];
                        for (int dim=0;dim<3;dim++){
                            rij[dim]=cudaCoords[3*particleNumber+dim]-cudaCoords[3*interactingParticle+dim];
                            if(rij[dim]>((cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
                            if(rij[dim]<(-(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2])/2)){rij[dim]=rij[dim]+(cudaBoxSize[dim*2+1]-cudaBoxSize[dim*2]);}
                            r=r+(rij[dim]*rij[dim]);
                        }
                        r=sqrtf(r);
/// carefull about reactions! !!!!!!!!!!!!!!!!!!!!!!!!!!
                        if(r<maxCutoff){
                            int bin = (int)(r*numberOfBins/maxCutoff);
                            atomicAdd(&cudaRDFMatrix[cudaTypes[particleNumber]*numberOfParticleTypes*numberOfBins+cudaTypes[interactingParticle]*numberOfBins+bin], 1);
                            atomicAdd(&cudaRDFMatrix[cudaTypes[interactingParticle]*numberOfParticleTypes*numberOfBins+cudaTypes[particleNumber]*numberOfBins+bin], 1);
                        }
                    }
                    interactingParticle=cudaNeighborList[2*interactingParticle+1];
                }
                while(interactingParticle!=cudaNeighborListBegins[todo[x]]);/// loop/list end
            }
        }
    }
}

__global__ void groups(float* cudaCoords, float* cudaForces, int* cudaTypes, int numberParticles, int numberOfParticleTypes, float * cudaCollisionRadiiMatrix, CudaOrderTwoPotential * cudaCudaGroupPotentials, GroupPart * cudaIndividualGroups, int numberOfIndividualGroups, float * cudaBoxSize){

    int pairNumber=blockIdx.x * blockDim.x + threadIdx.x;

    if(pairNumber<numberOfIndividualGroups){
        float r=0.0f;
        float rij[3];
        for (int dimension=0;dimension<3;dimension++){
            rij[dimension]=cudaCoords[3*cudaIndividualGroups[pairNumber].particle1+dimension]-cudaCoords[3*cudaIndividualGroups[pairNumber].particle2+dimension];
            if(rij[dimension]>((cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2])/2)){rij[dimension]=rij[dimension]-(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2]);}
            if(rij[dimension]<(-(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2])/2)){rij[dimension]=rij[dimension]+(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2]);}
            r=r+(rij[dimension]*rij[dimension]);
        }
        r=sqrtf(r);

        calculateOrderTwoPotential(cudaIndividualGroups[pairNumber].particle1, cudaIndividualGroups[pairNumber].particle2, cudaIndividualGroups[pairNumber].groupPot, r, cudaCoords, cudaForces, cudaTypes, cudaCudaGroupPotentials, numberOfParticleTypes, cudaCollisionRadiiMatrix);
    }
}

__global__ void update(float* cudaCoords, float* cudaForces, int* cudaTypes, float* cudaD, int * cudaNeighborList, int * cudaNeighborListBegins, float * cudaBoxSize, int * cudaSemaphore, hiprandState* globalRandStates, float dt, int numberParticles, float KB, float T, float maxCutoff, int * cudaLatticeSize){

    int particleNumber=blockIdx.x * blockDim.x + threadIdx.x;

    if(particleNumber<numberParticles){
        hiprandState localState = globalRandStates[particleNumber];

        int oldVoxel=   ((int)floor((cudaCoords[3*particleNumber+2]-cudaBoxSize[4])/maxCutoff)%cudaLatticeSize[2])*cudaLatticeSize[0]*cudaLatticeSize[1]
                        +((int)floor((cudaCoords[3*particleNumber+1]-cudaBoxSize[2])/maxCutoff)%cudaLatticeSize[1])*cudaLatticeSize[0]
                        +((int)floor((cudaCoords[3*particleNumber+0]-cudaBoxSize[0])/maxCutoff)%cudaLatticeSize[0]);

        for(int dimension=0; dimension<3; ++dimension){
            /// apply diffusion and forces -> update positions
            /// x(t+dt) = x(t) - dt*D*(F(x(t))/kT) + sqrt(2Ddt)*N(0,1)
            cudaCoords[particleNumber*3+dimension] += -dt*cudaD[cudaTypes[particleNumber]]*cudaForces[particleNumber*3+dimension]/KB/T + sqrt(2*cudaD[cudaTypes[particleNumber]]*dt)*hiprand_normal( &localState );
            //cudaCoords[particleNumber*3+dimension] += -dt*cudaD[cudaTypes[particleNumber]]*cudaForces[particleNumber*3+dimension]/KB/T ;
            //cudaCoords[particleNumber*3+dimension] += cudaForces[particleNumber*3+dimension] ;
            cudaForces[particleNumber*3+dimension]=0.0f;
            /// periodic boundary condition
            while(cudaCoords[3*particleNumber+dimension]>cudaBoxSize[dimension*2+1]){cudaCoords[3*particleNumber+dimension]=cudaCoords[3*particleNumber+dimension]-(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2+0]);}
            while(cudaCoords[3*particleNumber+dimension]<cudaBoxSize[dimension*2+0]){cudaCoords[3*particleNumber+dimension]=cudaCoords[3*particleNumber+dimension]+(cudaBoxSize[dimension*2+1]-cudaBoxSize[dimension*2+0]);}
        }
        /// lattice field changed?
        int newVoxel=   ((int)floor((cudaCoords[3*particleNumber+2]-cudaBoxSize[4])/maxCutoff)%cudaLatticeSize[2])*cudaLatticeSize[0]*cudaLatticeSize[1]
                    +((int)floor((cudaCoords[3*particleNumber+1]-cudaBoxSize[2])/maxCutoff)%cudaLatticeSize[1])*cudaLatticeSize[0]
                    +((int)floor((cudaCoords[3*particleNumber+0]-cudaBoxSize[0])/maxCutoff)%cudaLatticeSize[0]);

        /// apply voxel-changes ...
        if(newVoxel!=oldVoxel){
            bool leaveLoop = false;
            /// delete form old list
            while(!leaveLoop){
                /// Lock
                if(atomicExch(&(cudaSemaphore[oldVoxel]),1)==0){
                    int prev=cudaNeighborList[2*particleNumber];
                    int next=cudaNeighborList[2*particleNumber+1];
                    cudaNeighborList[2*prev+1]=next;
                    cudaNeighborList[2*next]=prev;
                    /// was this partilce begin of the linked list?
                    if(cudaNeighborListBegins[oldVoxel]==particleNumber){
                        /// was the particle the only one in this field?
                        if(cudaNeighborList[2*particleNumber]==particleNumber){
                            cudaNeighborListBegins[oldVoxel]=-1;
                        }
                        else{
                            cudaNeighborListBegins[oldVoxel]=cudaNeighborList[2*particleNumber+1];
                        }
                    }
                    leaveLoop=true;
                    /// unLock
                    atomicExch(&(cudaSemaphore[oldVoxel]),0);
                }
            }
            leaveLoop = false;
            /// push ontop of the new list
            while(!leaveLoop){
                /// Lock
                if(atomicExch(&(cudaSemaphore[newVoxel]),1)==0){
                    /// is new list empty?
                    if(cudaNeighborListBegins[newVoxel]!=-1){/// no
                        cudaNeighborList[2*particleNumber]=cudaNeighborList[2*cudaNeighborListBegins[newVoxel]];
                        cudaNeighborList[2*particleNumber+1]=cudaNeighborListBegins[newVoxel];
                        cudaNeighborList[2*cudaNeighborList[2*cudaNeighborListBegins[newVoxel]]+1]=particleNumber;;
                        cudaNeighborList[2*cudaNeighborListBegins[newVoxel]]=particleNumber;
                        cudaNeighborListBegins[newVoxel]=particleNumber;
                    }
                    else{/// first one in new list
                        cudaNeighborList[2*particleNumber+1]=particleNumber;
                        cudaNeighborList[2*particleNumber]=particleNumber;
                        cudaNeighborListBegins[newVoxel]=particleNumber;
                    }
                    leaveLoop=true;
                    /// unLock
                    atomicExch(&(cudaSemaphore[newVoxel]),0);
                }
            }
        }

        globalRandStates[particleNumber] = localState;
    }
    return;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed, int n ){
    int id=blockIdx.x * blockDim.x + threadIdx.x;
    if(id<n){
        hiprand_init ( seed, id, 0, &state[id] );
    }
}


/// pos force radii forceconst types todo links linkbegins
__device__ void lennardJones(){

    return;
}
